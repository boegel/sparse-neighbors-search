#include "hip/hip_runtime.h"
/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutor: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwigs-University Freiburg im Breisgau
**/

#include <stdio.h>
#include "sparseMatrix.h"
// #include <math.h>
#include "kernel.h"
// #include <hipcub/hipcub.hpp>
__device__ size_t computeHashValueCuda(size_t key, size_t aModulo) {
    // source:  Thomas Wang: Integer Hash Functions, 1997 / 2007 
    // https://gist.github.com/badboy/6267743
    key = key * A;
    key = ~key + (key << 15);
    key = key ^ (key >> 12);
    key = key + (key << 2);
    key = key ^ (key >> 4);
    key = key * 2057;
    key = key ^ (key >> 16);
    return key % aModulo;
}

__global__ void fitCuda(const size_t* pFeatureIdList, const size_t* pSizeOfInstanceList,
                    const size_t pNumberOfHashFunctions, const size_t pMaxNnz,
                    size_t* pComputedSignatures, 
                    const size_t pNumberOfInstances, const size_t pStartInstance, 
                    const size_t pBlockSize, const size_t pShingleSize,
                    size_t* pSignaturesBlockSize) {
                        
    int instanceId = blockIdx.x + pStartInstance;
    size_t minHashValue = MAX_VALUE;
    size_t hashValue = 0;
    size_t signatureSize = pNumberOfHashFunctions * pBlockSize / pShingleSize;
    int featureId = blockIdx.x * pMaxNnz;
    int hashFunctionId = threadIdx.x;
    size_t sizeOfInstance;
    size_t signatureBlockValue;
    size_t shingleId;
    size_t signatureBlockId = blockIdx.x * pNumberOfHashFunctions * pBlockSize;
    // compute one instance per block
    // if one instance is computed, block takes next instance
    while (instanceId < pNumberOfInstances) {
        // compute the minHashValue for every hash function
        // if pBlockSize is greater as 1, hash functions * pBlockSize values 
        // are computed. They will be merged together by a factor of pShingleSize
        sizeOfInstance = pSizeOfInstanceList[instanceId];
        while (hashFunctionId < pNumberOfHashFunctions * pBlockSize && featureId < pNumberOfInstances*pMaxNnz) {
            for (size_t i = 0; i < sizeOfInstance; ++i) {
                hashValue = computeHashValueCuda((pFeatureIdList[featureId + i]+1) * (hashFunctionId+1), MAX_VALUE);
                if (hashValue < minHashValue) {
                    minHashValue = hashValue;
                }
            }
            
            pSignaturesBlockSize[signatureBlockId + hashFunctionId] = minHashValue;
            hashFunctionId += blockDim.x;
            minHashValue = MAX_VALUE;
        }
        __syncthreads();
        // merge pShingleSize values together.
        // do one merge per thread
        hashFunctionId = threadIdx.x * pShingleSize;
        shingleId = threadIdx.x;
        while (hashFunctionId < pNumberOfHashFunctions * pBlockSize ) {
            signatureBlockValue = pSignaturesBlockSize[signatureBlockId + hashFunctionId];
            for (size_t i = 1; i < pShingleSize && hashFunctionId+i < pNumberOfHashFunctions * pBlockSize; ++i) {
                signatureBlockValue = computeHashValueCuda((pSignaturesBlockSize[signatureBlockId + hashFunctionId+i]+1) * (signatureBlockValue+1), MAX_VALUE);
            }
            pComputedSignatures[(instanceId-pStartInstance)*signatureSize + shingleId] = signatureBlockValue;
            hashFunctionId += blockDim.x * pShingleSize;
            shingleId += blockDim.x;
        }
        __syncthreads();
        instanceId += gridDim.x;
        featureId = instanceId * pMaxNnz;
        minHashValue = MAX_VALUE;
        hashFunctionId = threadIdx.x;
    }
}



__global__ void createSortedHistogramsCuda(size_t* pHitsPerInstance, size_t* pElementsPerInstance,
                                            const size_t pNumberOfInstances,
                                            int* pHistogram, size_t* pRadixSortMemory,
                                            size_t* pHistogramSortedWithId, 
                                            size_t* pNumberOfPossibleNeighbors,
                                            size_t pNumberOfNeighbors, size_t pExcessFactor,
                                            size_t* pNeighborhood, float* pDistances, size_t pFast) {
    // sort hits per instances
    // count instances
    // take highest pNeighborhood*excessfaktor + same hits count
    // to compute euclidean distance or cosine similarity
    
    // per block query one instance
    // sort these with the threads
    
    // printf("number of instances: %i", pNumberOfInstances);
    const int numberOfThreads = blockDim.x;
    // const int gridDim = gridDim.x;
    const int threadIdConst = threadIdx.x;
    
    int instanceId = blockIdx.x;
    int threadId = threadIdx.x;
    int startId = 0;
    int endId = 0;
    int startPositionSortingMemory = blockIdx.x * pNumberOfInstances * 2;
    int addValue = 1;
    size_t numberOfElementsToBeConsidered = pNumberOfNeighbors * pExcessFactor;
    // size_t index;
    // create histogram
    while (instanceId < pNumberOfInstances) {
        threadId = threadIdx.x;
        while (threadId < pNumberOfInstances) {
            // clear arrays to 0
            pHistogram[blockIdx.x * pNumberOfInstances + threadId] = 0;
            pHistogramSortedWithId[blockIdx.x * pNumberOfInstances*2 + threadId] = 0;
            pHistogramSortedWithId[blockIdx.x * pNumberOfInstances*2 + threadId+1] = 0;
            
            threadId += blockDim.x;
        }
        // instanceId += gridDim.x;
        // threadId = threadIdx.x;
        // __syncthreads();
        // continue;
        threadId = threadIdx.x;
        // compute start position in array pHitsPerInstance
        startId = instanceId;
        for (size_t i = 0; i < instanceId; ++i) {
            startId += pElementsPerInstance[i];
        }
        endId = startId+pElementsPerInstance[instanceId];
        // if (blockIdx.x == 0 && threadIdx.x == 0) {
        //     printf("\nstartId: %i", startId);
        //     printf("\nendId: %i\n", endId);
        //     return; 
        // }
        // start position for the histogram in the 1D array pHistogram
        // every block is having pNumberOfInstances positions
        size_t indexPosition = blockIdx.x*pNumberOfInstances;
        startId += threadIdx.x;
        while (startId < endId) {
            
            atomicAdd(&(pHistogram[indexPosition + pHitsPerInstance[startId]]), addValue);
            // instanceId += gridDim.x;
            startId += numberOfThreads;
        }
        
        // __syncthreads();
        // return;
        // instanceId += gridDim.x;
        // threadId = threadIdx.x;
        
        // continue;
        
        // if (instanceId == 0 && threadIdx.x == 0) {
        //     for (size_t i = 0; i < pNumberOfInstances; ++i) {
        //         printf("%i,", pHistogram[i]);
        //     }
        // }
        // __syncthreads();
        
        // position i --> number of hits
        // position i+1 --> instance id
        threadId = threadIdx.x;
        while (threadId < pNumberOfInstances) {
            // number of hits
            pHistogramSortedWithId[blockIdx.x * pNumberOfInstances*2 + threadId*2] = (size_t) pHistogram[blockIdx.x * pNumberOfInstances + threadId];
            // instance id
            pHistogramSortedWithId[blockIdx.x * pNumberOfInstances*2 + threadId*2 + 1] = (size_t) threadId;
            threadId += blockDim.x;  
        }
        __syncthreads(); 
        // return;
        radixSortDesc((size_t) blockIdx.x * pNumberOfInstances*2, pRadixSortMemory,
                        pHistogramSortedWithId, pNumberOfInstances);
        return;
        // count number of elements that should be considered in the euclidean distance 
        // or cosine similarity computation
        if (pFast) {
            // threadId = threadIdx.x;
            // insert the k neighbors and distances to the neighborhood and distances vector
            if (threadIdConst < pNumberOfNeighbors) {
                pNeighborhood[instanceId*pNumberOfNeighbors+threadIdConst] 
                    = pHistogramSortedWithId[startPositionSortingMemory + threadIdConst+1];
                pDistances[instanceId*pNumberOfNeighbors+threadIdConst] 
                    = (float) pHistogramSortedWithId[startPositionSortingMemory + threadIdConst];
            }
        } else {
            // excess factor is missing!!!
            
        }
    
        instanceId += gridDim.x;
        threadId = threadIdx.x;
        // startPositionSortingMemory += 
    }
}

__device__ void radixSortDesc(size_t pStartPosition, size_t* pRadixSortMemory,
                            size_t* pSortingMemory, size_t pNumberOfInstances) {
    // radix sort in descending order of the histogram
    // a[number_of_instances][0] == hits, [1] == elementID
    size_t threadId = threadIdx.x;
    size_t index = 0;
    int addValue = 1;
    size_t bucketNumber = 0;
    size_t nullIndexRadixSortNullBucket = blockIdx.x * pNumberOfInstances*2*2;
    size_t nullIndexRadixSortFirstBucket = blockIdx.x * pNumberOfInstances*2*2 + pNumberOfInstances*2;
    
    __shared__ int elementCount [2];
    // printf("\n\nfoo 211\n\n");
    if (threadIdx.x == 0) {
            elementCount[0] = 0;    
            elementCount[1] = 0; 
    }
    __syncthreads();
    
    // startPosition = blockIdx.x * pNumberOfInstances*2
    for (size_t i = 0; i < sizeof(size_t) * 8; ++i) {
        // partion phase: split numbers to bucket 0 or 1
        while (threadId < pNumberOfInstances) {
            bucketNumber = (pSortingMemory[pStartPosition+threadId*2] >> i) & 1;
            
            // printf("%i, ", bucketNumber);
            atomicAdd(&(elementCount[bucketNumber]), addValue);
            if (bucketNumber == 0) {
                index = nullIndexRadixSortNullBucket + threadId * 2;
            } else {
                index = nullIndexRadixSortFirstBucket + threadId * 2;
            }
            
            // index = pStartPosition+(bucketNumber*pNumberOfInstances*2) + threadId*2;
            pRadixSortMemory[index] = pSortingMemory[pStartPosition + threadId * 2];
            pRadixSortMemory[index+1] = pSortingMemory[pStartPosition + threadId * 2 + 1];
            threadId += blockDim.x;
        }
        __syncthreads();
        // if (blockIdx.x == 0 && threadIdx.x == 0) {
        //     printf("elementCount[0]: %i", elementCount[0]);
        //     printf("elementCount[1]: %i", elementCount[1]);
        // }
        
        // return;
        // collection phase copy values from the bucket 1 and then from bucket 0 to the array
        threadId = threadIdx.x;
        while (threadId < elementCount[1]) {
            index = nullIndexRadixSortFirstBucket + threadId*2;
            
            // index = pStartPosition + pNumberOfInstances*2 + threadId*2;
            pSortingMemory[pStartPosition + threadId*2] = pRadixSortMemory[index];
            pSortingMemory[pStartPosition + threadId*2+1] = pRadixSortMemory[index+1];
            pRadixSortMemory[index] = 0;
            pRadixSortMemory[index+1] = 0;
            threadId += blockDim.x;
        }
        
        threadId = threadIdx.x;
        size_t indexSortingMemory = pStartPosition + elementCount[1]*2;
        while (threadId < elementCount[0]) {
            // index = pStartPosition + threadId*2;
            index = nullIndexRadixSortNullBucket + threadId*2;
            
            pSortingMemory[indexSortingMemory + threadId*2] = pRadixSortMemory[index];
            pSortingMemory[indexSortingMemory + threadId*2+1] = pRadixSortMemory[index+1];
            pRadixSortMemory[index] = 0;
            pRadixSortMemory[index+1] = 0;
            threadId += blockDim.x;
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            elementCount[0] = 0;    
            elementCount[1] = 0; 
        }
        // if (blockIdx.x == 0 && threadIdx.x == 0) {
        //     printf("moooooo%i, \n\n", i);
        // }
        threadId = threadIdx.x;
        __syncthreads();
        // if (i == 1) {
        //     return;
        // }
    }
    // printf("\n\nfoo 250\n\n");
    
    
}
__device__ void radixSortAsc(size_t pStartPosition, size_t* pRadixSortMemory,
                            size_t* pSortingMemory, size_t pNumberOfInstances) {
    // radix sort in descending order of the histogram
    // a[number_of_instances][0] == hits, [1] == elementID
    size_t threadId = threadIdx.x * 2;
    size_t index = 0;
    int addValue = 1;
    size_t bucketNumber = 0;
    __shared__ int elementCount [2];
    for (int i = 0; i < sizeof(int) * 8; ++i) {
        // partion phase: split numbers to bucket 0 or 1
        while (threadId < pNumberOfInstances) {
            bucketNumber = (pSortingMemory[pStartPosition+threadId] >> i) & 1;
            atomicAdd(&(elementCount[bucketNumber]), addValue);
            index = pStartPosition+(bucketNumber*pNumberOfInstances) + threadId;
            pRadixSortMemory[index] =  pSortingMemory[pStartPosition + threadId];
            pRadixSortMemory[index+1] =  pSortingMemory[pStartPosition + threadId+1];
            threadId += blockDim.x;
        }
        __syncthreads();
        // collection phase copy values from the bucket 1 and then from bucket 0 to the array
        threadId = threadIdx.x * 2;
        while (threadId < pNumberOfInstances) {
            index = pStartPosition + threadId;
            pSortingMemory[index] = pRadixSortMemory[index];
            pSortingMemory[index+1] = pRadixSortMemory[index+1];
            threadId += blockDim.x;
        }
        
        threadId = threadIdx.x * 2;
        while (threadId < pNumberOfInstances) {
            index = pStartPosition + pNumberOfInstances + threadId;
            pSortingMemory[index] = pRadixSortMemory[index];
            pSortingMemory[index+1] = pRadixSortMemory[index+1];
            threadId += blockDim.x;
        }
        __syncthreads();
    }
}
__global__ void euclideanDistanceCuda(size_t* pHitsPerQueryInstance, size_t* pNumberInstancesToConsider, 
                                        size_t pRangeBetweenInstances, size_t pNumberOfInstances,
                                        size_t* pFeatureList, float* pValuesList,
                                        size_t* pSizeOfInstanceList, size_t pMaxNnz, 
                                        size_t* pRadixSortMemory, int pNumberOfNeighbors,
                                        size_t* pNeighborhood, float* pDistances) {
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;
    size_t pointerToFeatureInstance, pointerToFeatureNeighbor, queryIndexInstance,
        queryIndexNeighbor, instanceId, instanceIdNeighbor, indexSparseMatrixInstance,
        indexSparseMatrixNeighbor, numberOfFeaturesInstance, numberOfFeaturesNeighbor,
        featureIdNeighbor, featureIdInstance;
    bool endOfInstanceNotReached, endOfNeighborNotReached;
    float euclideanDistance, value;
    while (blockId < pNumberOfInstances) {
        // pointer to feature ids in sparse matrix
        pointerToFeatureInstance = 0;
        pointerToFeatureNeighbor = 0;
        
        // get the instance ids of the query instance and the possible neighbor
        // it is assumed that the first instance is the query instance and 
        // all others are possible neighbors
        queryIndexInstance = blockId * pRangeBetweenInstances;
        queryIndexNeighbor = blockId * pRangeBetweenInstances + threadId*2;
        
        // get the two instance ids
        instanceId = pHitsPerQueryInstance[queryIndexInstance+1];
        instanceIdNeighbor = pHitsPerQueryInstance[queryIndexNeighbor+1];
        
        // get the index positons for the two instances in the sparse matrix
        indexSparseMatrixInstance = instanceId*pMaxNnz;
        indexSparseMatrixNeighbor = instanceIdNeighbor*pMaxNnz;
        
        // get the number of features for every instance
        numberOfFeaturesInstance = pSizeOfInstanceList[instanceId];
        numberOfFeaturesNeighbor = pSizeOfInstanceList[instanceIdNeighbor];
        
        endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
        endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
        euclideanDistance = 0;
        value = 0;
        while (threadId < pNumberInstancesToConsider[instanceIdNeighbor]) {
            
            while (endOfInstanceNotReached && endOfNeighborNotReached) {
                featureIdInstance = pFeatureList[indexSparseMatrixInstance+pointerToFeatureInstance];
                featureIdNeighbor = pFeatureList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor];
                
                if (featureIdInstance == featureIdNeighbor) {
                    // if they are the same substract the values, compute the square and sum it up
                    value = pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance] 
                                    - pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor];
                    //this->getNextValue(pRowIdVector[i], pointerToMatrixElement) - queryData->getNextValue(pRowId, pointerToVectorElement);
                    euclideanDistance += value * value;
                    // increase both counters to the next element 
                    ++pointerToFeatureInstance;
                    ++pointerToFeatureNeighbor;
                } else if (featureIdInstance < featureIdNeighbor) {
                    // if the feature ids are unequal square only the smaller one and add it to the sum
                    value = pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance];
                    euclideanDistance += value * value;
                    // increase counter for first vector
                    ++pointerToFeatureInstance;
                } else {
                    value = pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor];
                    euclideanDistance += value * value;
                    ++pointerToFeatureNeighbor;
                }
                endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
                endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
            }
            while (endOfInstanceNotReached) {
                value = pValuesList[indexSparseMatrixInstance + pointerToFeatureInstance];
                euclideanDistance += value * value;
                ++pointerToFeatureInstance;
                endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
            }
            while (endOfNeighborNotReached) {
                value = pValuesList[indexSparseMatrixNeighbor + pointerToFeatureNeighbor];
                euclideanDistance += value * value;
                ++pointerToFeatureNeighbor;
                endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
            }
            
            // square root of the sum
            euclideanDistance = sqrtf(euclideanDistance);
            // store euclidean distance and neighbor id
            pHitsPerQueryInstance[queryIndexNeighbor] = (int) euclideanDistance * 1000;
            threadId += blockIdx.x;
            euclideanDistance = 0;
            value = 0;
            queryIndexNeighbor = blockId * pRangeBetweenInstances + threadId*2;
            instanceIdNeighbor = pHitsPerQueryInstance[queryIndexNeighbor + 1];
            indexSparseMatrixNeighbor = instanceIdNeighbor*pMaxNnz;
            numberOfFeaturesNeighbor = pSizeOfInstanceList[instanceIdNeighbor];
            pointerToFeatureInstance = 0;
            pointerToFeatureNeighbor = 0;
            endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
            endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
        }
        // sort instances by euclidean distance
        radixSortDesc(queryIndexInstance, 
                        pRadixSortMemory, pHitsPerQueryInstance, pNumberOfInstances);
        threadId = threadIdx.x;
        // insert the k neighbors and distances to the neighborhood and distances vector
        if (threadId < pNumberOfNeighbors) {
            pNeighborhood[instanceId*pNumberOfNeighbors+threadId] 
                = pHitsPerQueryInstance[queryIndexInstance + threadId+1];
            pDistances[instanceId*pNumberOfNeighbors+threadId] 
                = (float) pHitsPerQueryInstance[queryIndexInstance + threadId];
        }
        blockId += gridDim.x;
        threadId = threadIdx.x;
    }
    
}

__global__ void cosineSimilarityCuda(size_t* pHitsPerQueryInstance, size_t* pNumberInstancesToConsider, 
                                        size_t pRangeBetweenInstances, size_t pNumberOfInstances,
                                        size_t* pFeatureList, float* pValuesList,
                                        size_t* pSizeOfInstanceList, size_t pMaxNnz, 
                                        size_t* pRadixSortMemory, int pNumberOfNeighbors,
                                        size_t* pNeighborhood, float* pDistances) {
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;
    size_t pointerToFeatureInstance, pointerToFeatureNeighbor, queryIndexInstance,
        queryIndexNeighbor, instanceId, instanceIdNeighbor, indexSparseMatrixInstance,
        indexSparseMatrixNeighbor, numberOfFeaturesInstance, numberOfFeaturesNeighbor,
        featureIdNeighbor, featureIdInstance;
    bool endOfInstanceNotReached, endOfNeighborNotReached;
    float magnitudeInstance, magnitudeNeighbor, dotProduct, cosineSimilarity;
    while (blockId < pNumberOfInstances) {
        // pointer to feature ids in sparse matrix
        pointerToFeatureInstance = 0;
        pointerToFeatureNeighbor = 0;
        
        // get the instance ids of the query instance and the possible neighbor
        // it is assumed that the first instance is the query instance and 
        // all others are possible neighbors
        queryIndexInstance = blockId * pRangeBetweenInstances;
        queryIndexNeighbor = blockId * pRangeBetweenInstances + threadId*2;
        
        // get the two instance ids
        instanceId = pHitsPerQueryInstance[queryIndexInstance];
        instanceIdNeighbor = pHitsPerQueryInstance[queryIndexNeighbor];
        
        // get the index positons for the two instances in the sparse matrix
        indexSparseMatrixInstance = instanceId*pMaxNnz;
        indexSparseMatrixNeighbor = instanceIdNeighbor*pMaxNnz;
        
        // get the number of features for every instance
        numberOfFeaturesInstance = pSizeOfInstanceList[instanceId];
        numberOfFeaturesNeighbor = pSizeOfInstanceList[instanceIdNeighbor];
        
        endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
        endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
        magnitudeInstance = 0;
        magnitudeNeighbor = 0;
        dotProduct = 0;
        while (threadId < pNumberInstancesToConsider[instanceIdNeighbor]) {
            
            while (endOfInstanceNotReached && endOfNeighborNotReached) {
                featureIdInstance = pFeatureList[indexSparseMatrixInstance+pointerToFeatureInstance];
                featureIdNeighbor = pFeatureList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor];
                
                if (featureIdInstance == featureIdNeighbor) {
                    // if they are the same substract the values, compute the square and sum it up
                    dotProduct += pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance] 
                                    * pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor];
                    //this->getNextValue(pRowIdVector[i], pointerToMatrixElement) - queryData->getNextValue(pRowId, pointerToVectorElement);
                    magnitudeInstance += powf(pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance], 2);
                    magnitudeNeighbor += powf(pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor], 2);
                    // increase both counters to the next element 
                    ++pointerToFeatureInstance;
                    ++pointerToFeatureNeighbor;
                } else if (featureIdInstance < featureIdNeighbor) {
                    // if the feature ids are unequal square only the smaller one and add it to the sum
                    magnitudeInstance += powf(pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance], 2);
                    // increase counter for first vector
                    ++pointerToFeatureInstance;
                } else {
                    magnitudeNeighbor += powf(pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor], 2);
                    ++pointerToFeatureNeighbor;
                }
                endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
                endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
            }
            while (endOfInstanceNotReached) {
                magnitudeInstance += powf(pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance], 2);
                ++pointerToFeatureInstance;
                endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
            }
            while (endOfNeighborNotReached) {
                magnitudeNeighbor += powf(pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor], 2);
                ++pointerToFeatureNeighbor;
                endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
            }
            
            // square root of the sum
            cosineSimilarity = dotProduct / (float) magnitudeInstance * magnitudeNeighbor;
            // store euclidean distance and neighbor id
            pHitsPerQueryInstance[queryIndexNeighbor + 1] = (int) cosineSimilarity * 1000;
            threadId += blockIdx.x;
            magnitudeInstance = 0;
            magnitudeNeighbor = 0;
            dotProduct = 0;
            cosineSimilarity = 0;
            queryIndexNeighbor = blockId * pRangeBetweenInstances + threadId*2;
            instanceIdNeighbor = pHitsPerQueryInstance[queryIndexNeighbor];
            indexSparseMatrixNeighbor = instanceIdNeighbor*pMaxNnz;
            numberOfFeaturesNeighbor = pSizeOfInstanceList[instanceIdNeighbor];
            pointerToFeatureInstance = 0;
            pointerToFeatureNeighbor = 0;
            endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
            endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
        }
        // sort instances by euclidean distance
        radixSortDesc(queryIndexInstance, pRadixSortMemory, pHitsPerQueryInstance, pNumberOfInstances);
                        
        if (threadId < pNumberOfNeighbors) {
            pNeighborhood[instanceId*pNumberOfNeighbors+threadId] 
                = pHitsPerQueryInstance[queryIndexInstance + threadId+1];
            pDistances[instanceId*pNumberOfNeighbors+threadId] 
                = (float) pHitsPerQueryInstance[queryIndexInstance + threadId];
        }
        blockId += gridDim.x;
        threadId = threadIdx.x;
    }
    
}