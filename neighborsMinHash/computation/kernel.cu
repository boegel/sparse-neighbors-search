#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sparseMatrix.h"
#include <math.h>
#define float A = sqrt(2) - 1;

__global__ void fit(size_t* pFeatureIdList, size_t* pSizeOfInstanceList, size_t pNumberOfHashFunctions, size_t pMaxNnz
                        size_t* pComputedSignatures) {
    __shared__ size_t signature[pNumberOfHashFunctions];
    size_t threadId = threadIdx.x;
    size_t blockId = blockIdx.x;
    size_t minHashValue = MAX_VALUE;
    size_t hashValue = 0;
    if (blockId < pRawData->size()) {
        // size_t sizeOfInstance = pSizeOfInstanceList[blockId];
        for (size_t i = 0; i < pSizeOfInstanceList[blockId]; ++i) {
            hashValue = computeHashValue((pFeatureIdList[blockId*pMaxNnz + i] + 1)*threadId, MAX_VALUE);
            if (hashValue < minHashValue) {
                minHashValue = hashValue;
            }
        }
        signature[threadId] = minHashValue;
    }
    __syncthread();
    
    if (threadId == 0) {
        for (size_t i = 0; i < pNumberOfHashFunctions; ++i) {
            pComputedSignatures[blockId*pNumberOfHashFunctions +i] = signature[i];
        }
    }
}

__device__ size_t computeHashValue(size_t key, size_t aModulo) {
    // source:  Thomas Wang: Integer Hash Functions, 1997 / 2007 
    // https://gist.github.com/badboy/6267743
    key = key * A;
    key = ~key + (key << 15);
    key = key ^ (key >> 12);
    key = key + (key << 2);
    key = key ^ (key >> 4);
    key = key * 2057;
    key = key ^ (key >> 16);
    return key % aModulo;
}

__global__ void query(size_t** pSignature, size_t** pInverseIndex, size_t* pElementsPerHashFunction, size_t pNumberOfHashFunctions,
                        size_t* pHitsToBeReturned) {
    size_t threadId = threadIdx.x;
    size_t blockId = blockIdx.x;
    size_t hashValue = pSignature[blockId][threadId];
    __shared__ size_t hits[pNumberOfHashFunctions];
    
    for (size_t i = 0; i < pElementsPerHashFunction[threadId]; ++i) {
        if (hashValue == pInverseIndex[threadId][i]) {
            hits[threadId] = i;
        } else if (hashValue > pInverseIndex[threadId][i]) {
            hits[threadId] = MAX_VALUE;
            continue;
        }
    }
    __syncthread();
    
    if (threadId == 0) {
        for (size_t i = 0; i < pNumberOfHashFunctions; ++i) {
            pHitsToBeReturned[blockId*pNumberOfHashFunctions + i] = hits[i];
        }
    }  
}

__global__ void euclidianDistance(size_t* pFeatureIds, size_t* pSizeOfInstanceList,
                                    float* pFeatureValues, size_t pMaxNnz,
                                    size_t* pPossibleInstances, size_t* pSizePerInstance, size_t pMaxCandidates,                             
                                    size_t* pHitsToBeReturned, float* pValuesToBeReturned) {
    size_t threadId = threadIdx.x;
    size_t blockId = blockIdx.x;
    // size_t hashValue = pSignature[blockId][threadId];
    // __shared__ size_t hits[pNumberOfHashFunctions];
    size_t startIndexInstance = *  
    size_t endIndexInstance = 
    
}

__global__ void cosineSimilarity() {
    
}