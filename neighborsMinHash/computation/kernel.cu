#include "hip/hip_runtime.h"
/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutor: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwigs-University Freiburg im Breisgau
**/

#include <stdio.h>
#include "sparseMatrix.h"
// #include <math.h>
#include "kernel.h"
// #include <hipcub/hipcub.hpp>
__device__ size_t computeHashValueCuda(size_t key, size_t aModulo) {
    // source:  Thomas Wang: Integer Hash Functions, 1997 / 2007 
    // https://gist.github.com/badboy/6267743
    key = key * A;
    key = ~key + (key << 15);
    key = key ^ (key >> 12);
    key = key + (key << 2);
    key = key ^ (key >> 4);
    key = key * 2057;
    key = key ^ (key >> 16);
    return key % aModulo;
}

__global__ void fitCuda(const size_t* pFeatureIdList, const size_t* pSizeOfInstanceList,
                    const size_t pNumberOfHashFunctions, const size_t pMaxNnz,
                    size_t* pComputedSignatures, 
                    const size_t pNumberOfInstances, const size_t pStartInstance, 
                    const size_t pBlockSize, const size_t pShingleSize,
                    size_t* pSignaturesBlockSize) {
                        
    int instanceId = blockIdx.x + pStartInstance;
    size_t minHashValue = MAX_VALUE;
    size_t hashValue = 0;
    size_t signatureSize = pNumberOfHashFunctions * pBlockSize / pShingleSize;
    int featureId = blockIdx.x * pMaxNnz;
    int hashFunctionId = threadIdx.x;
    size_t sizeOfInstance;
    size_t signatureBlockValue;
    size_t shingleId;
    size_t signatureBlockId = blockIdx.x * pNumberOfHashFunctions * pBlockSize;
    // compute one instance per block
    // if one instance is computed, block takes next instance
    while (instanceId < pNumberOfInstances) {
        // compute the minHashValue for every hash function
        // if pBlockSize is greater as 1, hash functions * pBlockSize values 
        // are computed. They will be merged together by a factor of pShingleSize
        sizeOfInstance = pSizeOfInstanceList[instanceId];
        while (hashFunctionId < pNumberOfHashFunctions * pBlockSize && featureId < pNumberOfInstances*pMaxNnz) {
            for (size_t i = 0; i < sizeOfInstance; ++i) {
                hashValue = computeHashValueCuda((pFeatureIdList[featureId + i]+1) * (hashFunctionId+1), MAX_VALUE);
                if (hashValue < minHashValue) {
                    minHashValue = hashValue;
                }
            }
            
            pSignaturesBlockSize[signatureBlockId + hashFunctionId] = minHashValue;
            hashFunctionId += blockDim.x;
            minHashValue = MAX_VALUE;
        }
        __syncthreads();
        // merge pShingleSize values together.
        // do one merge per thread
        hashFunctionId = threadIdx.x * pShingleSize;
        shingleId = threadIdx.x;
        while (hashFunctionId < pNumberOfHashFunctions * pBlockSize ) {
            signatureBlockValue = pSignaturesBlockSize[signatureBlockId + hashFunctionId];
            for (size_t i = 1; i < pShingleSize && hashFunctionId+i < pNumberOfHashFunctions * pBlockSize; ++i) {
                signatureBlockValue = computeHashValueCuda((pSignaturesBlockSize[signatureBlockId + hashFunctionId+i]+1) * (signatureBlockValue+1), MAX_VALUE);
            }
            pComputedSignatures[(instanceId-pStartInstance)*signatureSize + shingleId] = signatureBlockValue;
            hashFunctionId += blockDim.x * pShingleSize;
            shingleId += blockDim.x;
        }
        __syncthreads();
        instanceId += gridDim.x;
        featureId = instanceId * pMaxNnz;
        minHashValue = MAX_VALUE;
        hashFunctionId = threadIdx.x;
    }
}



__global__ void queryCuda(size_t* pHitsPerInstance, size_t* pSizePerInstance,
                            size_t pNeighborhoodSize, size_t* pNeighborhood,
                            float* pDistances, const size_t pNumberOfInstances,
                            int* pHistogramMemory, int* pRadixSortMemory,
                            int* pSortedInstancesByNumberOfHits, int* pNumberOfPossibleNeighbors,
                            size_t pNumberOfNeighbors, size_t pExcessFactor) {
    // sort hits per instances
    // count instances
    // take highest pNeighborhood*excessfaktor + same hits count
    // to compute euclidean distance or cosine similarity
    
    
    // per block query one instance
    // sort these with the threads
    
    
    const int numberOfThreads = blockDim.x;
    int instanceId = blockIdx.x;
    int threadId = threadIdx.x;
    int startId;
    int endId;
    int startPositionSortingMemory = blockIdx.x * pNumberOfInstances * 2;
    int addValue = 1;
    size_t numberOfElementsToBeConsidered = pNumberOfNeighbors * pExcessFactor;
    // size_t index;
    // create histogram
    while (instanceId < pNumberOfInstances) {
        for (size_t i = 0; i < pNumberOfInstances; ++i) {
            // clear arrays to 0
            pHistogramMemory[instanceId * pNumberOfInstances + i] = 0;
        }
        // compute start position in array pHitsPerInstance
        startId = instanceId;
        for (size_t i = 0; i < instanceId; ++i) {
            startId += pSizePerInstance[i];
        }
        endId = startId+pSizePerInstance[instanceId];
        
        while (startId + threadId < endId) {
            atomicAdd(&(pHistogramMemory[pHitsPerInstance[startId+threadId] * instanceId]), addValue);
            instanceId += gridDim.x;
            threadId += numberOfThreads;
        }
        
        __syncthreads();
        threadId = threadIdx.x;
        while (threadId < pNumberOfInstances) {
            pSortedInstancesByNumberOfHits[startPositionSortingMemory + threadId] = pHistogramMemory[startId+threadId];
            pSortedInstancesByNumberOfHits[startPositionSortingMemory + threadId + 1] = startId+threadId;
            threadId += blockDim.x;  
        }
        __syncthreads();

        radixSortDesc(startPositionSortingMemory, MAX_VALUE, pRadixSortMemory,
                        pSortedInstancesByNumberOfHits, pNumberOfInstances);
        
        // count number of elements that should be considered in the euclidean distance 
        // or cosine similarity computation
        
        if (pSortedInstancesByNumberOfHits[startPositionSortingMemory + numberOfElementsToBeConsidered*2] != 0) {
            // pSortedInstancesByNumberOfHits[startPositionSortingMemory + numberOfElementsToBeConsidered*2] != 0
        }
    
        instanceId += gridDim.x;
        threadId = threadIdx.x;
    }
}

__device__ void radixSortDesc(int pStartPosition, int pEndPosition, int* pRadixSortMemory,
                            int* pSortingMemory, size_t pNumberOfInstances) {
    // radix sort in descending order of the histogram
    // a[number_of_instances][0] == hits, [1] == elementID
    size_t threadId = threadIdx.x * 2;
    size_t index = 0;
    int addValue = 1;
    size_t bucketNumber = 0;
    __shared__ int elementCount [2];
    for (int i = 0; i < sizeof(int) * 8; ++i) {
        // partion phase: split numbers to bucket 0 or 1
        while (threadId < pNumberOfInstances && threadId < pEndPosition) {
            bucketNumber = (pSortingMemory[pStartPosition+threadId] >> i) & 1;
            atomicAdd(&(elementCount[bucketNumber]), addValue);
            index = pStartPosition+(bucketNumber*pNumberOfInstances) + threadId;
            pRadixSortMemory[index] =  pSortingMemory[pStartPosition + threadId];
            pRadixSortMemory[index+1] =  pSortingMemory[pStartPosition + threadId+1];
            threadId += blockDim.x;
        }
        __syncthreads();
        // collection phase copy values from the bucket 1 and then from bucket 0 to the array
        threadId = threadIdx.x * 2;
        while (threadId < pNumberOfInstances && threadId < pEndPosition) {
            index = pStartPosition + pNumberOfInstances + threadId;
            pSortingMemory[index] = pRadixSortMemory[index];
            pSortingMemory[index+1] = pRadixSortMemory[index+1];
            threadId += blockDim.x;
        }
        
        threadId = threadIdx.x * 2;
        while (threadId < pNumberOfInstances && threadId < pEndPosition) {
            index = pStartPosition + threadId;
            pSortingMemory[index] = pRadixSortMemory[index];
            pSortingMemory[index+1] = pRadixSortMemory[index+1];
            threadId += blockDim.x;
        }
        __syncthreads();
    }
}
__device__ void radixSortAsc(int pStartPosition, int pEndPosition, int* pRadixSortMemory,
                            int* pSortingMemory, size_t pNumberOfInstances) {
    // radix sort in descending order of the histogram
    // a[number_of_instances][0] == hits, [1] == elementID
    size_t threadId = threadIdx.x * 2;
    size_t index = 0;
    int addValue = 1;
    size_t bucketNumber = 0;
    __shared__ int elementCount [2];
    for (int i = 0; i < sizeof(int) * 8; ++i) {
        // partion phase: split numbers to bucket 0 or 1
        while (threadId < pNumberOfInstances && threadId < pEndPosition) {
            bucketNumber = (pSortingMemory[pStartPosition+threadId] >> i) & 1;
            atomicAdd(&(elementCount[bucketNumber]), addValue);
            index = pStartPosition+(bucketNumber*pNumberOfInstances) + threadId;
            pRadixSortMemory[index] =  pSortingMemory[pStartPosition + threadId];
            pRadixSortMemory[index+1] =  pSortingMemory[pStartPosition + threadId+1];
            threadId += blockDim.x;
        }
        __syncthreads();
        // collection phase copy values from the bucket 1 and then from bucket 0 to the array
        threadId = threadIdx.x * 2;
        while (threadId < pNumberOfInstances && threadId < pEndPosition) {
            index = pStartPosition + threadId;
            pSortingMemory[index] = pRadixSortMemory[index];
            pSortingMemory[index+1] = pRadixSortMemory[index+1];
            threadId += blockDim.x;
        }
        
        threadId = threadIdx.x * 2;
        while (threadId < pNumberOfInstances && threadId < pEndPosition) {
            index = pStartPosition + pNumberOfInstances + threadId;
            pSortingMemory[index] = pRadixSortMemory[index];
            pSortingMemory[index+1] = pRadixSortMemory[index+1];
            threadId += blockDim.x;
        }
        __syncthreads();
    }
}
__global__ void euclidianDistanceCuda(int* pHitsPerQueryInstance, int* pNumberInstancesToConsider, 
                                        size_t pRangeBetweenInstances, size_t pNumberOfInstances,
                                        size_t* pFeatureList, float* pValuesList,
                                        size_t* pSizeOfInstanceList, size_t pMaxNnz, 
                                        int* pRadixSortMemory, int pNumberOfNeighbors) {
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;
    size_t pointerToFeatureInstance, pointerToFeatureNeighbor, queryIndexInstance,
        queryIndexNeighbor, instanceId, instanceIdNeighbor, indexSparseMatrixInstance,
        indexSparseMatrixNeighbor, numberOfFeaturesInstance, numberOfFeaturesNeighbor,
        featureIdNeighbor, featureIdInstance;
    bool endOfInstanceNotReached, endOfNeighborNotReached;
    float euclideanDistance, value;
    while (blockId < pNumberOfInstances) {
        // pointer to feature ids in sparse matrix
        pointerToFeatureInstance = 0;
        pointerToFeatureNeighbor = 0;
        
        // get the instance ids of the query instance and the possible neighbor
        // it is assumed that the first instance is the query instance and 
        // all others are possible neighbors
        queryIndexInstance = blockId * pRangeBetweenInstances;
        queryIndexNeighbor = blockId * pRangeBetweenInstances + threadId*2;
        
        // get the two instance ids
        instanceId = pHitsPerQueryInstance[queryIndexInstance];
        instanceIdNeighbor = pHitsPerQueryInstance[queryIndexNeighbor];
        
        // get the index positons for the two instances in the sparse matrix
        indexSparseMatrixInstance = instanceId*pMaxNnz;
        indexSparseMatrixNeighbor = instanceIdNeighbor*pMaxNnz;
        
        // get the number of features for every instance
        numberOfFeaturesInstance = pSizeOfInstanceList[instanceId];
        numberOfFeaturesNeighbor = pSizeOfInstanceList[instanceIdNeighbor];
        
        endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
        endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
        euclideanDistance = 0;
        value = 0;
        while (threadId < pInstancesToConsider[instanceIdNeighbor]) {
            
            while (endOfInstanceNotReached && endOfNeighborNotReached) {
                featureIdInstance = pFeatureList[indexSparseMatrixInstance+pointerToFeatureInstance];
                featureIdNeighbor = pFeatureList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor];
                
                if (featureIdInstance == featureIdNeighbor) {
                    // if they are the same substract the values, compute the square and sum it up
                    value = pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance] 
                                    - pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor];
                    //this->getNextValue(pRowIdVector[i], pointerToMatrixElement) - queryData->getNextValue(pRowId, pointerToVectorElement);
                    euclideanDistance += value * value;
                    // increase both counters to the next element 
                    ++pointerToFeatureInstance;
                    ++pointerToFeatureNeighbor;
                } else if (featureIdInstance < featureIdNeighbor) {
                    // if the feature ids are unequal square only the smaller one and add it to the sum
                    value = pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance];
                    euclideanDistance += value * value;
                    // increase counter for first vector
                    ++pointerToFeatureInstance;
                } else {
                    value = pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor];
                    euclideanDistance += value * value;
                    ++pointerToFeatureNeighbor;
                }
                endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
                endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
            }
            while (endOfInstanceNotReached) {
                value = pValuesList[indexSparseMatrixInstance + pointerToFeatureInstance];
                euclideanDistance += value * value;
                ++pointerToFeatureInstance;
                endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
            }
            while (endOfNeighborNotReached) {
                value = pValuesList[indexSparseMatrixNeighbor + pointerToFeatureNeighbor];
                euclideanDistance += value * value;
                ++pointerToFeatureNeighbor;
                endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
            }
            
            // square root of the sum
            euclideanDistance = sqrtf(euclideanDistance);
            // store euclidean distance and neighbor id
            pHitsPerQueryInstance[queryIndexNeighbor + 1] = (int) euclideanDistance * 1000;
            threadId += blockIdx.x;
            euclideanDistance = 0;
            value = 0;
            queryIndexNeighbor = blockId * pRangeBetweenInstances + threadId*2;
            instanceIdNeighbor = pHitsPerQueryInstance[queryIndexNeighbor];
            indexSparseMatrixNeighbor = instanceIdNeighbor*pMaxNnz;
            numberOfFeaturesNeighbor = pSizeOfInstanceList[instanceIdNeighbor];
            pointerToFeatureInstance = 0;
            pointerToFeatureNeighbor = 0;
            endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
            endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
        }
        // sort instances by euclidean distance
        radixSortDesc(queryIndexInstance, pInstancesToConsider[instanceId], 
                        pRadixSortMemory, pHitsPerQueryInstance, pNumberOfInstances);
        blockId += gridDim.x;
        threadId = threadIdx.x;
    }
    
}

__global__ void cosineSimilarityCuda(int* pHitsPerQueryInstance, int* pInstancesToConsider, 
                                        size_t pRangeBetweenInstances, size_t pNumberOfInstances,
                                        size_t* pFeatureList, float* pValuesList,
                                        size_t* pSizeOfInstanceList, size_t pMaxNnz, 
                                        int* pRadixSortMemory, int pNumberOfNeighbors) {
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;
    size_t pointerToFeatureInstance, pointerToFeatureNeighbor, queryIndexInstance,
        queryIndexNeighbor, instanceId, instanceIdNeighbor, indexSparseMatrixInstance,
        indexSparseMatrixNeighbor, numberOfFeaturesInstance, numberOfFeaturesNeighbor,
        featureIdNeighbor, featureIdInstance;
    bool endOfInstanceNotReached, endOfNeighborNotReached;
    float magnitudeInstance, magnitudeNeighbor, dotProduct, cosineSimilarity;
    while (blockId < pNumberOfInstances) {
        // pointer to feature ids in sparse matrix
        pointerToFeatureInstance = 0;
        pointerToFeatureNeighbor = 0;
        
        // get the instance ids of the query instance and the possible neighbor
        // it is assumed that the first instance is the query instance and 
        // all others are possible neighbors
        queryIndexInstance = blockId * pRangeBetweenInstances;
        queryIndexNeighbor = blockId * pRangeBetweenInstances + threadId*2;
        
        // get the two instance ids
        instanceId = pHitsPerQueryInstance[queryIndexInstance];
        instanceIdNeighbor = pHitsPerQueryInstance[queryIndexNeighbor];
        
        // get the index positons for the two instances in the sparse matrix
        indexSparseMatrixInstance = instanceId*pMaxNnz;
        indexSparseMatrixNeighbor = instanceIdNeighbor*pMaxNnz;
        
        // get the number of features for every instance
        numberOfFeaturesInstance = pSizeOfInstanceList[instanceId];
        numberOfFeaturesNeighbor = pSizeOfInstanceList[instanceIdNeighbor];
        
        endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
        endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
        magnitudeInstance = 0;
        magnitudeNeighbor = 0;
        dotProduct = 0;
        while (threadId < pInstancesToConsider[instanceIdNeighbor]) {
            
            while (endOfInstanceNotReached && endOfNeighborNotReached) {
                featureIdInstance = pFeatureList[indexSparseMatrixInstance+pointerToFeatureInstance];
                featureIdNeighbor = pFeatureList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor];
                
                if (featureIdInstance == featureIdNeighbor) {
                    // if they are the same substract the values, compute the square and sum it up
                    dotProduct += pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance] 
                                    * pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor];
                    //this->getNextValue(pRowIdVector[i], pointerToMatrixElement) - queryData->getNextValue(pRowId, pointerToVectorElement);
                    magnitudeInstance += powf(pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance], 2);
                    magnitudeNeighbor += powf(pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor], 2);
                    // increase both counters to the next element 
                    ++pointerToFeatureInstance;
                    ++pointerToFeatureNeighbor;
                } else if (featureIdInstance < featureIdNeighbor) {
                    // if the feature ids are unequal square only the smaller one and add it to the sum
                    magnitudeInstance += powf(pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance], 2);
                    // increase counter for first vector
                    ++pointerToFeatureInstance;
                } else {
                    magnitudeNeighbor += powf(pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor], 2);
                    ++pointerToFeatureNeighbor;
                }
                endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
                endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
            }
            while (endOfInstanceNotReached) {
                magnitudeInstance += powf(pValuesList[indexSparseMatrixInstance+pointerToFeatureInstance], 2);
                ++pointerToFeatureInstance;
                endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
            }
            while (endOfNeighborNotReached) {
                magnitudeNeighbor += powf(pValuesList[indexSparseMatrixNeighbor+pointerToFeatureNeighbor], 2);
                ++pointerToFeatureNeighbor;
                endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
            }
            
            // square root of the sum
            cosineSimilarity = dotProduct / (float) magnitudeInstance * magnitudeNeighbor;
            // store euclidean distance and neighbor id
            pHitsPerQueryInstance[queryIndexNeighbor + 1] = (int) cosineSimilarity * 1000;
            threadId += blockIdx.x;
            magnitudeInstance = 0;
            magnitudeNeighbor = 0;
            dotProduct = 0;
            cosineSimilarity = 0;
            queryIndexNeighbor = blockId * pRangeBetweenInstances + threadId*2;
            instanceIdNeighbor = pHitsPerQueryInstance[queryIndexNeighbor];
            indexSparseMatrixNeighbor = instanceIdNeighbor*pMaxNnz;
            numberOfFeaturesNeighbor = pSizeOfInstanceList[instanceIdNeighbor];
            pointerToFeatureInstance = 0;
            pointerToFeatureNeighbor = 0;
            endOfInstanceNotReached = pointerToFeatureInstance < numberOfFeaturesInstance;
            endOfNeighborNotReached = pointerToFeatureNeighbor < numberOfFeaturesNeighbor;
        }
        // sort instances by euclidean distance
        radixSortDesc(queryIndexInstance, pInstancesToConsider[instanceId],
                        pRadixSortMemory, pHitsPerQueryInstance, pNumberOfInstances);
        blockId += gridDim.x;
        threadId = threadIdx.x;
    }
    
}