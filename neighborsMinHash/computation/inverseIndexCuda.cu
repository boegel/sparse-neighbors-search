#include "hip/hip_runtime.h"
/**
 Copyright 2015 Joachim Wolff
 Master Thesis
 Tutors: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwig-University Freiburg im Breisgau
**/

#include <algorithm>
#include <iostream>
#include <iterator>
#include <utility>

#ifdef OPENMP
#include <omp.h>
#endif

#include "inverseIndexCuda.h"
// #include "kSizeSortedMap.h"
#include "kernel.h"


// class sort_map {
//   public:
//     size_t key;
//     size_t val;
// };

// bool mapSortDescByValue(const sort_map& a, const sort_map& b) {
//         return a.val > b.val;
// };

InverseIndexCuda::InverseIndexCuda(size_t pNumberOfHashFunctions, size_t pShingleSize,
                    size_t pNumberOfCores, size_t pChunkSize,
                    size_t pMaxBinSize, size_t pMinimalBlocksInCommon,
                    size_t pExcessFactor, size_t pMaximalNumberOfHashCollisions, size_t pBloomierFilter,
                    int pPruneInverseIndex, float pPruneInverseIndexAfterInstance, int pRemoveHashFunctionWithLessEntriesAs,
                    size_t pHashAlgorithm, size_t pBlockSize, size_t pShingle, size_t pRemoveValueWithLeastSigificantBit):InverseIndex() {   
        // std::cout << __LINE__ << std::endl;
                        
    mNumberOfHashFunctions = pNumberOfHashFunctions;
    mShingleSize = pShingleSize;
    mNumberOfCores = pNumberOfCores;
    mChunkSize = pChunkSize;
    mMaxBinSize = pMaxBinSize;
    mMinimalBlocksInCommon = pMinimalBlocksInCommon;
    mExcessFactor = pExcessFactor;
    mMaximalNumberOfHashCollisions = pMaximalNumberOfHashCollisions;
    mPruneInverseIndex = pPruneInverseIndex;
    mPruneInverseIndexAfterInstance = pPruneInverseIndexAfterInstance;
    mRemoveHashFunctionWithLessEntriesAs = pRemoveHashFunctionWithLessEntriesAs;
    mHashAlgorithm = pHashAlgorithm;
    mBlockSize = pBlockSize;
    mShingle = pShingle;
    size_t inverseIndexSize;
    if (mShingle == 0) {
        if (mBlockSize == 0) {
            mBlockSize = 1;
        }
        inverseIndexSize = mNumberOfHashFunctions * mBlockSize;
    } else {
        inverseIndexSize = ceil(((float) (mNumberOfHashFunctions * mBlockSize) / (float) mShingleSize));        
    }
    
    
}
 
InverseIndexCuda::~InverseIndexCuda() {
   hipFree(mDev_FeatureList);
   hipFree(mDev_ComputedSignaturesPerInstance);
   hipFree(mDev_SizeOfInstanceList);
  
}

distributionInverseIndex* InverseIndexCuda::getDistribution() {
    return mInverseIndexStorage->getDistribution();
}

 // compute the signature for one instance
vsize_t InverseIndexCuda::computeSignature(const SparseMatrixFloat* pRawData, const size_t pInstance) {

   
}

vsize_t InverseIndexCuda::shingle(vsize_t pSignature) {
  

}

vsize_t InverseIndexCuda::computeSignatureWTA(const SparseMatrixFloat* pRawData, const size_t pInstance) {
    
          
}

umap_uniqueElement* InverseIndexCuda::computeSignatureMap(const SparseMatrixFloat* pRawData) {
}
void InverseIndexCuda::fit(const SparseMatrixFloat* pRawData) {
    printf("foo");
    int maxBlocks = 65535;
    printf("number of hash functions: %i", mNumberOfHashFunctions);
    if (mNumberOfHashFunctions % 32 != 0) {
        mNumberOfHashFunctions += 32 - (mNumberOfHashFunctions % 32);
    }
    printf("number of hash functions: %i", mNumberOfHashFunctions);
    
    hipMalloc((void **) &mDev_FeatureList,
               pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t));
    hipMalloc((void **) &mDev_SizeOfInstanceList,
               pRawData->getNumberOfInstances() * sizeof(size_t));
    hipMalloc((void **) &mDev_ComputedSignaturesPerInstance,
               pRawData->getNumberOfInstances()* mNumberOfHashFunctions * sizeof(size_t));
    hipMemcpy(mDev_FeatureList, pRawData->getSparseMatrixIndex(),
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t),
               hipMemcpyHostToDevice);
    hipMemcpy(mDev_SizeOfInstanceList, pRawData->getSparseMatrixSizeOfInstances(),
            pRawData->getNumberOfInstances() * sizeof(size_t),
            hipMemcpyHostToDevice);
    
    // fitGpu<<<pRawData->getNumberOfInstances(), mNumberOfHashFunctions, mNumberOfHashFunctions>>>
   
    fitCuda<<<128, 128, mNumberOfHashFunctions>>>
    (mDev_FeatureList, 
    mDev_SizeOfInstanceList, 
    mNumberOfHashFunctions, 
    pRawData->getMaxNnz(),
            mDev_ComputedSignaturesPerInstance, 
            pRawData->getNumberOfInstances());
    size_t* instancesHashValues = (size_t*) malloc(pRawData->getNumberOfInstances() * mNumberOfHashFunctions * sizeof(size_t));
    hipMemcpy(instancesHashValues, mDev_ComputedSignaturesPerInstance, 
                pRawData->getNumberOfInstances() * mNumberOfHashFunctions * sizeof(size_t),
                hipMemcpyDeviceToHost);
   for(size_t i = 0; i < pRawData->getNumberOfInstances(); ++i) {
       printf("Instance: %i of %i", i, pRawData->getNumberOfInstances());
       for (size_t j = 0; j < mNumberOfHashFunctions; ++j) {
           printf("%i,", instancesHashValues[i*mNumberOfHashFunctions + j]);
       }
       printf("\n");
   }
   free(instancesHashValues);
}

neighborhood* InverseIndexCuda::kneighbors(const umap_uniqueElement* pSignaturesMap, 
                                        const size_t pNneighborhood, const bool pDoubleElementsStorageCount) {
// compute hits in the inverse index on the gpu and 
// return a list with all the associated index values per hash function
// process them on the gpu,
// compute exact neighbors based on the hits on gpu.

}