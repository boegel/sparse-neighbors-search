#include "hip/hip_runtime.h"
/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutors: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwigs-University Freiburg im Breisgau
**/

#include "inverseIndexCuda.h"
#include "kernel.h"

InverseIndexCuda::InverseIndexCuda(size_t pNumberOfHashFunctions, 
                                    size_t pShingle, size_t pShingleSize, 
                                    size_t pBlockSize) {
          mNumberOfHashFunctions = pNumberOfHashFunctions;
          mShingle = pShingle;
          mShingleSize = pShingleSize;
          mBlockSize = pBlockSize;
}
InverseIndexCuda::~InverseIndexCuda() {
    hipFree(mDev_FeatureList);
    hipFree(mDev_ValuesList);
    hipFree(mDev_SizeOfInstanceList);
}
void InverseIndexCuda::copyDataToGpu(const SparseMatrixFloat* pRawData) {
    // memory for instances and their featureIds
    hipMalloc((void **) &mDev_FeatureList,
            pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t));
    // memory for the values of the features of the instances
    hipMalloc((void **) &mDev_ValuesList, 
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(float));
    // memory for the number of features per instance
    hipMalloc((void **) &mDev_SizeOfInstanceList,
            pRawData->getNumberOfInstances() * sizeof(size_t));
    
    // copy instances and their feature ids to the gpu
    hipMemcpy(mDev_FeatureList, pRawData->getSparseMatrixIndex(),
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t),
            hipMemcpyHostToDevice);
    // copy instances and their values for each feature to the gpu
    hipMemcpy(mDev_ValuesList, pRawData->getSparseMatrixValues(),
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(float),
            hipMemcpyHostToDevice);
    // copy the size of all instances to the gpu               
    hipMemcpy(mDev_SizeOfInstanceList, pRawData->getSparseMatrixSizeOfInstances(),
            pRawData->getNumberOfInstances() * sizeof(size_t),
            hipMemcpyHostToDevice);
}
void InverseIndexCuda::computeSignaturesOnGpu(const SparseMatrixFloat* pRawData, 
                                                size_t pStartIndex, size_t pEndIndex, 
                                                size_t pNumberOfInstances, size_t pNumberOfBlocks, 
                                                size_t pNumberOfThreads, size_t pShingleFactor, 
                                                size_t pBlockSizeShingle,
                                                vvsize_t_p* pSignatures) {
    // check if enough memory is available on the gpu 
    size_t memory_total = 0;
    size_t memory_free = 0;
    size_t iterations = 1;
    hipMemGetInfo(&memory_free, &memory_total);
    if (memory_free >= pRawData->getNumberOfInstances()  * mNumberOfHashFunctions * sizeof(size_t)) {
        iterations = ceil(pRawData->getNumberOfInstances()  * mNumberOfHashFunctions * sizeof(size_t) / static_cast<float>(memory_free));
    }
    
    size_t start = 0;
    size_t end = pRawData->getNumberOfInstances() / iterations;
    size_t windowSize = pRawData->getNumberOfInstances() / iterations;
    size_t* instancesHashValues = (size_t*) malloc(pRawData->getNumberOfInstances() / iterations * mNumberOfHashFunctions * sizeof(size_t));
    
    size_t signaturesSize = mNumberOfHashFunctions * pBlockSizeShingle / pShingleFactor;
    // memory for the inverse index on the gpu.
    // for each instance the number of hash functions
    hipMalloc((void **) &mDev_ComputedSignaturesPerInstance,
            pRawData->getNumberOfInstances() / iterations  * signaturesSize * sizeof(size_t));
    size_t* dev_SignaturesBlockSize;
    hipMalloc((void **) &dev_SignaturesBlockSize,
           pNumberOfBlocks * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(size_t));
    // compute the signatures on the gpu
    // do it in n iterations with equal sized chunks 
    // if the data would not fit on the ram of the gpu
    for (size_t i = 0; i < iterations; ++i) {
        // execute kernel on gpu
        fitCuda<<<pNumberOfBlocks, pNumberOfThreads>>>
        (mDev_FeatureList, 
        mDev_SizeOfInstanceList,  
        mNumberOfHashFunctions, 
        pRawData->getMaxNnz(),
                mDev_ComputedSignaturesPerInstance, 
                end, start, mBlockSize, mShingleSize, dev_SignaturesBlockSize);
        // copy results back to host      
        hipMemcpy(instancesHashValues, mDev_ComputedSignaturesPerInstance, 
                    pRawData->getNumberOfInstances()/iterations * signaturesSize * sizeof(size_t),
                    hipMemcpyDeviceToHost);
        // copy values into one vector per instance
        for(size_t i = start; i < end; ++i) {
            vsize_t* instance = new vsize_t(signaturesSize);
            for (size_t j = 0; j < signaturesSize; ++j) {
                (*instance)[j] = instancesHashValues[i*signaturesSize + j];
            }
            (*pSignatures)[i] = instance;
        }
        
        start = end+1;
        end = end + windowSize;
    }
    
    hipFree(mDev_ComputedSignaturesPerInstance);
    hipFree(dev_SignaturesBlockSize);
}

void InverseIndexCuda::computeHitsOnGpu(std::vector<vvsize_t_p*>* pHitsPerInstance, 
                                                neighborhood* pNeighborhood, 
                                                size_t pNeighborhoodSize,
                                                size_t pNumberOfInstances,
                                                size_t pNumberOfBlocks) {
    vsize_t hitsPerInstance;
    vsize_t sizePerInstance;
    size_t counter = 0;
    
    for (auto it = pHitsPerInstance->begin(); it != pHitsPerInstance->end(); ++it) {
        for (auto itQueryInstance = (*it)->begin(); itQueryInstance != (*it)->end(); ++itQueryInstance) {
            for(auto itInstance = (*itQueryInstance)->begin(); 
                itInstance != (*itQueryInstance)->end(); ++itInstance) {
                    hitsPerInstance.push_back(*itInstance);
                    ++counter;
            }
        }
        sizePerInstance.push_back(counter);
        counter = 0;
    }
    size_t* dev_HitsPerInstances;
    size_t* dev_SizePerInstances;
    int* dev_HistogramMemory;
    int* dev_SortingMemory;
    int* dev_RadixSortMemory;
    // std::cout << "Size of hitPerInstnace: " << hitsPerInstance.size() << std::endl;
    hipMalloc((void **) &dev_HitsPerInstances,
            hitsPerInstance.size() * sizeof(size_t));
    hipMalloc((void **) &dev_SizePerInstances,
            sizePerInstance.size() * sizeof(size_t));
    hipMalloc((void **) &dev_HistogramMemory,
            pNumberOfBlocks*pNumberOfInstances * sizeof(int));
    hipMalloc((void **) &dev_SortingMemory,
            pNumberOfBlocks * pNumberOfInstances * 2 * sizeof(int));
    hipMalloc((void **) &dev_RadixSortMemory,
            pNumberOfBlocks * pNumberOfInstances * 2 * 2 * sizeof(int));
    hipMemcpy(dev_HitsPerInstances, &hitsPerInstance[0],
                hitsPerInstance.size() * sizeof(size_t),
            hipMemcpyHostToDevice);
    hipMemcpy(dev_SizePerInstances, &sizePerInstance[0],
                sizePerInstance.size() * sizeof(size_t),
            hipMemcpyHostToDevice);
    size_t* dev_Neighborhood;
    float* dev_Distances;
    int* neighborhood = (int*) malloc(pHitsPerInstance->size() * pNeighborhoodSize * sizeof(int));
    float* distances = (float*) malloc(pHitsPerInstance->size() * pNeighborhoodSize * sizeof(float));
    
    // size_t* instancesHashValues = (size_t*) malloc(pRawData->getNumberOfInstances() / iterations * mNumberOfHashFunctions * sizeof(size_t));
    
    hipMalloc((void **) &dev_Neighborhood,
                pHitsPerInstance->size() * pNeighborhoodSize * sizeof(size_t));
    hipMalloc((void **) &dev_Distances,
                pHitsPerInstance->size() * pNeighborhoodSize * sizeof(float));
    
    queryCuda<<<pNumberOfBlocks, pNumberOfInstances>>>
                (dev_HitsPerInstances, dev_SizePerInstances,
                pNeighborhoodSize, dev_Neighborhood,
                dev_Distances, pHitsPerInstance->size(), dev_HistogramMemory,
                dev_RadixSortMemory, dev_SortingMemory);
    
    if (!pFast) {
        if (pDistance) {
            euclideanDistanceCuda<<<512, 32>>>(dev_SortingMemory,
            );
        } else {
            cosineSimilarityCuda<<<512, 32>>>(dev_SortingMemory,
            );
        }
    }
    
    hipMemcpy(neighborhood, dev_Neighborhood,
                pHitsPerInstance->size() * pNeighborhoodSize * sizeof(int),
                hipMemcpyDeviceToHost);
    hipMemcpy(distances, dev_Distances,
                pHitsPerInstance->size() * pNeighborhoodSize * sizeof(float),
                hipMemcpyDeviceToHost);
                
     hipFree(dev_Neighborhood);
     hipFree(dev_Distances);
     // transfer to neighorhood layout
     vvint* neighborsVector = new vvint(pHitsPerInstance->size());
     vvfloat* distancesVector = new vvfloat(pHitsPerInstance->size());
     
     for (size_t i = 0; i < pHitsPerInstance->size(); ++i) {
         vint neighbors_;
         vfloat distances_;
         for (size_t j = 0; j < pNeighborhoodSize; ++j) {
             neighbors_.push_back(neighborhood[i*pNeighborhoodSize + j]);
             distances_.push_back(distances[i*pNeighborhoodSize + j]);
         }
         (*neighborsVector)[i] = neighbors_;
         (*distancesVector)[i] = distances_;
     }
     
     pNeighborhood->neighbors = neighborsVector;
     pNeighborhood->distances = distancesVector;
     free(neighborhood);
     free(distances);
     
     // return it
     // delete memory
     // 
     // check if everything is fitting in gpu memory,
     // loop if not.              
}
