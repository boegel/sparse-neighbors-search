#include "hip/hip_runtime.h"
/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutors: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwig-University Freiburg im Breisgau
**/

#include <algorithm>
#include <iostream>
#include <iterator>
#include <utility>

#ifdef OPENMP
#include <omp.h>
#endif

#include "inverseIndex.h"
#include "kSizeSortedMap.h"
#include "kernel.h"

class sort_map {
  public:
    size_t key;
    size_t val;
};

bool mapSortDescByValue(const sort_map& a, const sort_map& b) {
        return a.val > b.val;
};
InverseIndex::InverseIndex(){};
InverseIndex::InverseIndex(size_t pNumberOfHashFunctions, size_t pShingleSize,
                    size_t pNumberOfCores, size_t pChunkSize,
                    size_t pMaxBinSize, size_t pMinimalBlocksInCommon,
                    size_t pExcessFactor, size_t pMaximalNumberOfHashCollisions, size_t pBloomierFilter,
                    int pPruneInverseIndex, float pPruneInverseIndexAfterInstance, int pRemoveHashFunctionWithLessEntriesAs,
                    size_t pHashAlgorithm, size_t pBlockSize, size_t pShingle, size_t pRemoveValueWithLeastSigificantBit) {   
    mNumberOfHashFunctions = pNumberOfHashFunctions;
    mShingleSize = pShingleSize;
    mNumberOfCores = pNumberOfCores;
    mChunkSize = pChunkSize;
    mMaxBinSize = pMaxBinSize;
    mMinimalBlocksInCommon = pMinimalBlocksInCommon;
    mExcessFactor = pExcessFactor;
    mMaximalNumberOfHashCollisions = pMaximalNumberOfHashCollisions;
    mPruneInverseIndex = pPruneInverseIndex;
    mPruneInverseIndexAfterInstance = pPruneInverseIndexAfterInstance;
    mRemoveHashFunctionWithLessEntriesAs = pRemoveHashFunctionWithLessEntriesAs;
    mHashAlgorithm = pHashAlgorithm;
    mSignatureStorage = new umap_uniqueElement();
    mHash = new Hash();
    mBlockSize = pBlockSize;
    mShingle = pShingle;
    if (mShingle == 0) {
        if (mBlockSize == 0) {
            mBlockSize = 1;
        }
        mInverseIndexSize = mNumberOfHashFunctions * mBlockSize;
    } else {
        mInverseIndexSize = ceil(((float) (mNumberOfHashFunctions * mBlockSize) / (float) mShingleSize));        
    }
        mInverseIndexStorage = new InverseIndexStorageUnorderedMap(mInverseIndexSize, mMaxBinSize);
    mRemoveValueWithLeastSigificantBit = pRemoveValueWithLeastSigificantBit;
}
 
InverseIndex::~InverseIndex() {
    delete mSignatureStorage;
    delete mHash;
    delete mInverseIndexStorage;
}

distributionInverseIndex* InverseIndex::getDistribution() {
    return mInverseIndexStorage->getDistribution();
}

 // compute the signature for one instance
vsize_t* InverseIndex::computeSignature(const SparseMatrixFloat* pRawData, const size_t pInstance) {
    vsize_t* signature = new vsize_t(mNumberOfHashFunctions * mBlockSize);

    for(size_t j = 0; j < mNumberOfHashFunctions * mBlockSize; ++j) {
            size_t minHashValue = MAX_VALUE;        
            for (size_t i = 0; i < pRawData->getSizeOfInstance(pInstance); ++i) {
                size_t hashValue = mHash->hash((pRawData->getNextElement(pInstance, i) +1), (j+1), MAX_VALUE);
                if (hashValue < minHashValue) {
                    minHashValue = hashValue;
                }
            }
            (*signature)[j] = minHashValue;
    }
    // reduce number of hash values by a factor of mShingleSize
    if (mShingle) {
        return shingle(signature);
    }
    return signature;
}

vsize_t* InverseIndex::shingle(vsize_t* pSignature) {
    
    vsize_t* signature = new vsize_t(mNumberOfHashFunctions*mBlockSize / mShingleSize);
    
    if (mShingle == 1) {
        
        // if 0 than combine hash values inside the block to one new hash value
        size_t signatureBlockValue;
        size_t count = 0;
        
        for (size_t i = 0; i < signature->size(); ++i) {
            if (i*mShingleSize >= pSignature->size()) break;
            signatureBlockValue = (*pSignature)[i*mShingleSize];
            
            for (size_t j = 1; j < mShingleSize; ++j) {
                signatureBlockValue = mHash->hash((*pSignature)[i*mShingleSize+j]+1, signatureBlockValue+1, MAX_VALUE);
            }
            (*signature)[i] = signatureBlockValue;
            count = i;
        }
        
        signatureBlockValue = (*pSignature)[count*mShingleSize];
        for (size_t j = count; count * mShingleSize + j < pSignature->size(); ++j) {
            signatureBlockValue = mHash->hash((*pSignature)[count * mShingleSize + j]+1, signatureBlockValue+1, MAX_VALUE);
        }
        
        (*signature)[count+1] = signatureBlockValue;
        
    } else if (mShingle == 2) {
        // if 1 than take the minimum hash values of that block as the hash value
        // size_t k = 0;
        
        // while (k < mNumberOfHashFunctions*mBlockSize) {
        // // use computed hash value as a seed for the next computation
        //     size_t minValue = MAX_VALUE;
        //     for (size_t j = 0; j < mShingleSize  && k+j < mNumberOfHashFunctions*mBlockSize; ++j) {
        //         if (minValue > pSignature[k+j] ) {
        //             minValue = pSignature[k+j];
        //         }
        //     }
        //     signature->push_back(minValue);
        //     k += mShingleSize; 
        // }
    }
    delete pSignature;
    return signature; 
}

vsize_t* InverseIndex::computeSignatureWTA(const SparseMatrixFloat* pRawData, const size_t pInstance) {
    size_t sizeOfInstance = pRawData->getSizeOfInstance(pInstance);
    
    size_t mSeed = 42;
    size_t mK = mBlockSize;
    
    vsize_t* signature = new vsize_t (mNumberOfHashFunctions);;
    if (sizeOfInstance < mK) {
        mK = sizeOfInstance;
    }
    KSizeSortedMap keyValue(mK);
    
    for (size_t i = 0; i < mNumberOfHashFunctions; ++i) {
        
        for (size_t j = 0; j < sizeOfInstance; ++j) {
            size_t hashIndex = mHash->hash((pRawData->getNextElement(pInstance, j) +1), mSeed+i, MAX_VALUE);
            keyValue.insert(hashIndex, pRawData->getNextValue(pInstance, j));
        } 
        
        float maxValue = 0.0;
        size_t maxValueIndex = 0;
        for (size_t j = 0; j < mK; ++j) {
            if (keyValue.getValue(j) > maxValue) {
                maxValue = keyValue.getValue(j);
                maxValueIndex = j;
            }
        }
        (*signature)[i] = maxValueIndex;
        keyValue.clear();
    }
    if (mShingle) {
        return shingle(signature);
    }
    return signature;
}

umap_uniqueElement* InverseIndex::computeSignatureMap(const SparseMatrixFloat* pRawData) {
    mDoubleElementsQueryCount = 0;
    const size_t sizeOfInstances = pRawData->size();
    umap_uniqueElement* instanceSignature = new umap_uniqueElement();
    instanceSignature->reserve(sizeOfInstances);
    if (mChunkSize <= 0) {
        mChunkSize = ceil(pRawData->size() / static_cast<float>(mNumberOfCores));
    }
#ifdef OPENMP
    omp_set_dynamic(0);
#endif

#ifdef OPENMP
#pragma omp parallel for schedule(static, mChunkSize) num_threads(mNumberOfCores)
#endif
    for(size_t index = 0; index < pRawData->size(); ++index) {
        // compute unique id
        size_t signatureId = 0;
        for (size_t j = 0; j < pRawData->getSizeOfInstance(index); ++j) {
                signatureId = mHash->hash((pRawData->getNextElement(index, j) +1), (signatureId+1), MAX_VALUE);
        }
        // signature is in storage && 
        auto signatureIt = (*mSignatureStorage).find(signatureId);
        if (signatureIt != (*mSignatureStorage).end() && (instanceSignature->find(signatureId) != instanceSignature->end())) {
#ifdef OPENMP
#pragma omp critical
#endif
            {
                (*instanceSignature)[signatureId] = (*mSignatureStorage)[signatureId];
                (*instanceSignature)[signatureId].instances->push_back(index);
                mDoubleElementsQueryCount += (*mSignatureStorage)[signatureId].instances->size();
            }
            continue;
        }

        // for every hash function: compute the hash values of all features and take the minimum of these
        // as the hash value for one hash function --> h_j(x) = argmin (x_i of x) f_j(x_i)
        vsize_t* signature;
        if (mHashAlgorithm == 0) {
            // use minHash
            signature = computeSignature(pRawData, index);
        } else if (mHashAlgorithm == 1) {
            // use wta hash
            signature = computeSignatureWTA(pRawData, index);
        }
#ifdef OPENMP
#pragma omp critical
#endif
        {
            if (instanceSignature->find(signatureId) == instanceSignature->end()) {
                vsize_t* doubleInstanceVector = new vsize_t(1);
                (*doubleInstanceVector)[0] = index;
                uniqueElement element;
                element.instances = doubleInstanceVector; 
                element.signature = signature;
                (*instanceSignature)[signatureId] = element;
            } else {
                (*instanceSignature)[signatureId].instances->push_back(index);
                mDoubleElementsQueryCount += 1;
            } 
        }
    }
    return instanceSignature;
}
void InverseIndex::fit(const SparseMatrixFloat* pRawData) {
    std::cout << __LINE__ << std::endl;
    size_t pruneEveryNIterations = pRawData->size() * mPruneInverseIndexAfterInstance;
    size_t pruneCount = 0;
    mDoubleElementsStorageCount = 0;
    if (mChunkSize <= 0) { 
        mChunkSize = ceil(pRawData->size() / static_cast<float>(mNumberOfCores));
    }
    std::cout << __LINE__ << std::endl;
    
#ifdef OPENMP
    
#endif
#ifndef OPENMP
    // mNumberOfCores = 1;
#endif
    vvsize_t_p signatures;
    omp_set_nested(1);
    omp_set_dynamic(0);
    // omp_set_num_threads(mNumberOfCores);
    // omp_set_nested(1);
    
#pragma omp parallel num_threads(mNumberOfCores)
    {
        vvsize_t_p signaturesPerThread;
        // (pRawData->size() / mNumberOfCores);
        // size_t substractFactor = omp_get_thread_num() - 1 * pRawData->size() / 2 / mNumberOfCores;
        #pragma omp master nowait
        {
            // memory for instances and their featureIds
            hipMalloc((void **) &mDev_FeatureList,
                    pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t));
            // memory for the values of the features of the instances
            hipMalloc((void **) &mDev_ValuesList, 
                        pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(float));
            // memory for the number of features per instance
            hipMalloc((void **) &mDev_SizeOfInstanceList,
                    pRawData->getNumberOfInstances() * sizeof(size_t));
            
            // copy instances and their feature ids to the gpu
            hipMemcpy(mDev_FeatureList, pRawData->getSparseMatrixIndex(),
                        pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t),
                    hipMemcpyHostToDevice);
            // copy instances and their values for each feature to the gpu
            hipMemcpy(mDev_ValuesList, pRawData->getSparseMatrixValues(),
                        pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(float),
                    hipMemcpyHostToDevice);
            // copy the size of all instances to the gpu               
            hipMemcpy(mDev_SizeOfInstanceList, pRawData->getSparseMatrixSizeOfInstances(),
                    pRawData->getNumberOfInstances() * sizeof(size_t),
                    hipMemcpyHostToDevice);
            
            // check if enough memory is available on the gpu 
            size_t memory_total = 0;
            size_t memory_free = 0;
            size_t iterations = 1;
            hipMemGetInfo(&memory_free, &memory_total);
            std::cout << "memory total: " << memory_total << " memory free: " << memory_free << std::endl;
            std::cout << "sizeof)size_t) : " << sizeof(size_t) << std::endl;
            std::cout << "Needed memory: " << pRawData->getNumberOfInstances()  * mNumberOfHashFunctions * sizeof(size_t) << std::endl;
            if (memory_free >= pRawData->getNumberOfInstances()  * mNumberOfHashFunctions * sizeof(size_t)) {
                iterations = ceil(pRawData->getNumberOfInstances()  * mNumberOfHashFunctions * sizeof(size_t) / static_cast<float>(memory_free));
            }
            std::cout << "Iterations: " << iterations << std::endl;
            size_t start = 0;
            size_t end = pRawData->getNumberOfInstances() / iterations;
            size_t windowSize = pRawData->getNumberOfInstances() / iterations;
            size_t* instancesHashValues = (size_t*) malloc(pRawData->getNumberOfInstances() / iterations * mNumberOfHashFunctions * sizeof(size_t));
            std::cout << __LINE__ << std::endl;
            
            // memory for the inverse index on the gpu.
            // for each instance the number of hash functions
            hipMalloc((void **) &mDev_ComputedSignaturesPerInstance,
                    pRawData->getNumberOfInstances() / iterations  * mNumberOfHashFunctions * sizeof(size_t));
            
            for (size_t i = 0; i < iterations; ++i) {
                
                fitCuda<<<64, 128, mNumberOfHashFunctions * sizeof(size_t)>>>
                (mDev_FeatureList, 
                mDev_SizeOfInstanceList,  
                mNumberOfHashFunctions, 
                pRawData->getMaxNnz(),
                        mDev_ComputedSignaturesPerInstance, 
                        end, start);
                std::cout << __LINE__ << std::endl;
                        
                hipMemcpy(instancesHashValues, mDev_ComputedSignaturesPerInstance, 
                            pRawData->getNumberOfInstances()/iterations * mNumberOfHashFunctions * sizeof(size_t),
                            hipMemcpyDeviceToHost);
                std::cout << __LINE__ << std::endl;
                            
                for(size_t i = 0; i < pRawData->getNumberOfInstances() / iterations; ++i) {
                    // printf("Instance: %zu of %zu: ", i, pRawData->getNumberOfInstances());
                    vsize_t* instance = new vsize_t(mNumberOfHashFunctions);
                    for (size_t j = 0; j < mNumberOfHashFunctions; ++j) {
                        (*instance)[j] = instancesHashValues[i*mNumberOfHashFunctions + j];
                        // printf("%zu,", instancesHashValues[i*mNumberOfHashFunctions + j]);
                    }
                    signaturesPerThread.push_back(instance);
                    // printf("\n");
                }
                
                #pragma omp critical
                signatures.insert(signatures.end(), signaturesPerThread.begin(), signaturesPerThread.end());
                signaturesPerThread.clear();
                start = end+1;
                end = end + windowSize;
            }
            hipFree(mDev_ComputedSignaturesPerInstance);
            
        }
       
        // std::cout << __LINE__ << std::endl;
        // mInverseIndexStorage->reserveSpaceForMaps(pRawData->size() / 2);
        // for (size_t i = 0; i < signatures.size(); ++i) {
        //     // std::cout << "i: " << i << std::endl;
        //     for (size_t j = 0; j < signatures[i]->size(); ++j) {
        //         // std::cout << signatures[i]->operator[](j) << ",";
                
        //         mInverseIndexStorage->insert(j, signatures[i]->operator[](j), i, mRemoveValueWithLeastSigificantBit);
        //     }
        //     // std::cout << std::endl;
        // }
    #pragma omp for nowait schedule(static)
            for (size_t instance = pRawData->size() / 2; instance < pRawData->size(); ++instance) {
                std::cout << "for no wait thread id: " << omp_get_thread_num() << ", ";
                // if (omp_get_thread_num() == 0) break;
                if (mHashAlgorithm == 0) {
                    // use minHash
                    signaturesPerThread.push_back(computeSignature(pRawData, instance));
                    
                } else if (mHashAlgorithm == 1) {
                    // use wta hash
                    signaturesPerThread.push_back(computeSignatureWTA(pRawData, instance));
                }
            }
                
      #pragma omp for schedule(static) ordered
            for(int i=0; i < omp_get_num_threads(); i++) {
            #pragma omp ordered
                signatures.insert(signatures.end(), signaturesPerThread.begin(), signaturesPerThread.end());
            }       
        
    } 
// #ifndef OPENMP
//     signatures = signaturesPerThread;
// #endif

    // for (size_t i = 0; i < signatures.size(); ++i) {
    //     std::cout << "Instance " << i << " foo size: "<< signatures[i]->size() << std::endl;
    //     for (size_t j = 0; j < signatures[i]->size(); ++j) {
    //         std::cout << (signatures[i])->operator[](j) << ", ";
    //     }
    //     std::cout << std::endl;
    // }
    
    // std::cout << "Size of signatue: " << signatures.size() << std::endl;

    // std::cout << __LINE__ << std::endl;
    // return;
    // add gpu part
    // merge gpu signatures with cpu signatures
    
//     std::cout << "insert to inverse index: " << std::endl;
// // create inverse index 
// #ifdef OPENMP
//     omp_set_dynamic(0);
// #endif
// #ifdef OPENMP
// #pragma omp parallel num_threads(1)
// #endif
//     { 
//     // std::cout << __LINE__ << std::endl;
        
//          vector__umapVector_ptr inverseIndex (mInverseIndexSize);
//         //  size_t substractFactor = omp_get_thread_num() * (mInverseIndexSize / mNumberOfCores);
         
//          for (size_t i = 0; i < mInverseIndexSize; ++i) {
//              inverseIndex[i] = new umapVector_ptr();
//              inverseIndex[i]->reserve(pRawData->size() / 2);
//          }
//     // std::cout << __LINE__ << std::endl;
         
         
// #ifdef OPENMP
// #pragma omp for  
// #endif

//         for (size_t i = 0; i < mInverseIndexSize; ++i) {
//             for (size_t j = 0; j < signatures.size(); ++j) {
//                 size_t hashValue = (signatures[j])->operator[](i);
//                 if (mRemoveValueWithLeastSigificantBit) {
//                     size_t leastSignificantBits = 0b11111111111111111111111111111111 << mRemoveValueWithLeastSigificantBit;
//                     size_t insertValue = hashValue | leastSignificantBits;
//                     if (insertValue == leastSignificantBits) {
//                         continue;
//                     }
//                 }       
//                 auto itHashValue_InstanceVector = inverseIndex[i]->find(hashValue);

//                 // if for hash function h_i() the given hash values is already stored
//                 if (itHashValue_InstanceVector != inverseIndex[i]->end()) {
//                     // insert the instance id if not too many collisions (maxBinSize)
//                     if (itHashValue_InstanceVector->second->size() && itHashValue_InstanceVector->second->size() < mMaxBinSize) {
//                         // insert only if there wasn't any collisions in the past
//                         if (itHashValue_InstanceVector->second->size() > 0) {
//                             itHashValue_InstanceVector->second->push_back(j);
//                         }
//                     } else { 
//                         // too many collisions: delete stored ids. empty vector is interpreted as an error code 
//                         // for too many collisions
//                         itHashValue_InstanceVector->second->clear();
//                     }
//                 } else {
//                     // given hash value for the specific hash function was not avaible: insert new hash value
//                     vsize_t* instanceIdVector = new vsize_t(1);
//                     (*instanceIdVector)[0] = j;
//                     inverseIndex[i]->operator[](hashValue) = instanceIdVector;
//                 }       
//             }
//         }

// #ifdef OPENMP
// #pragma omp for schedule(static) ordered
//         for(int i=0; i<omp_get_num_threads(); i++) {
//             #pragma omp ordered
//             mInverseIndexStorage->insert(inverseIndex.begin(), inverseIndex.end());
//         }       
// #endif
//     } 
    
    // for (size_t i = 0; i < mInverseIndexStorage->size(); ++i) {
    //     std::cout << "hash function: " << i << std::endl;
    //     for (auto it = mInverseIndexStorage->getIndex()->operator[](i)->begin();
    //             it != mInverseIndexStorage->getIndex()->operator[](i)->end(); ++it) {
    //                 std::cout << "hashValue: " << it->first << ": ";
    //                 for (auto itVec = it->second->begin(); itVec != it->second->end(); ++itVec) {
    //                     std::cout << *itVec << ",";
    //                 }
    //                 std::cout << std::endl;
    //             }
    //                 std::cout << std::endl;
                
    // }
    
// #ifdef OPENMP
//     omp_set_dynamic(0);
// #endif
// #ifdef OPENMP
// #pragma omp parallel for schedule(static, mChunkSize) num_threads(mNumberOfCores)
// #endif
//     for (size_t index = 0; index < pRawData->size(); ++index) {
//         size_t signatureId = 0;
//         for (size_t j = 0; j < pRawData->getSizeOfInstance(index); ++j) {
//             signatureId = mHash->hash((pRawData->getNextElement(index, j) +1), (signatureId+1), MAX_VALUE);
//         }
//         vsize_t* signature;
//         auto itSignatureStorage = mSignatureStorage->find(signatureId);
//         if (itSignatureStorage == mSignatureStorage->end()) {
//             if (mHashAlgorithm == 0) {
//                 // use minHash
//                 signature = computeSignature(pRawData, index);
//             } else if (mHashAlgorithm == 1) {
//                 // use wta hash
//                 signature = computeSignatureWTA(pRawData, index);
//             }
//         } else {
//             signature = itSignatureStorage->second.signature;
//         }
// #ifdef OPENMP
// #pragma omp critical
// #endif
//         {   
//             ++pruneCount;
//             if (itSignatureStorage == mSignatureStorage->end()) {
//                 vsize_t* doubleInstanceVector = new vsize_t(1);
//                 (*doubleInstanceVector)[0] = index;
//                 uniqueElement element;
//                 element.instances = doubleInstanceVector;
//                 element.signature = signature;
//                 mSignatureStorage->operator[](signatureId) = element;
//             } else {
//                  mSignatureStorage->operator[](signatureId).instances->push_back(index);
//                  mDoubleElementsStorageCount += 1;
//             }
//         }
        
//         for (size_t j = 0; j < signature->size(); ++j) {
//             mInverseIndexStorage->insert(j, (*signature)[j], index, mRemoveValueWithLeastSigificantBit);
//         }
        
//         if (mPruneInverseIndexAfterInstance > 0) {
// #ifdef OPENMP
// #pragma omp critical
// #endif
//             {
//                 if (pruneCount >= pruneEveryNIterations) {
//                     pruneCount = 0;
                    
//                     if (mPruneInverseIndex > 0) {
//                         mInverseIndexStorage->prune(static_cast<size_t>(mPruneInverseIndex));
//                     }
//                     if (mRemoveHashFunctionWithLessEntriesAs >= 0) {
//                         mInverseIndexStorage->removeHashFunctionWithLessEntriesAs(static_cast<size_t>(mRemoveHashFunctionWithLessEntriesAs));
//                     }
//                 }
//             }           
//         }
//     }
//     	std::cout << __LINE__ << std::endl;
    
//     if (mPruneInverseIndex > 0) {
//         mInverseIndexStorage->prune(mPruneInverseIndex);
//     }
//     	std::cout << __LINE__ << std::endl;
    
//     if (mRemoveHashFunctionWithLessEntriesAs >= 0) {
//         mInverseIndexStorage->removeHashFunctionWithLessEntriesAs(static_cast<size_t>(mRemoveHashFunctionWithLessEntriesAs));
//     }
//     // for (std::cout << )
//     	std::cout << __LINE__ << std::endl;
//         std::cout << "Number of hash function: " << mInverseIndexStorage->size() << std::endl;
//         for (size_t i = 0; i < mInverseIndexStorage->size(); ++i) {
//             std::cout << "hash function: " << i << " Size: " << mInverseIndexStorage->getIndex()->operator[](i)->size();
//             std::cout << " Load factor: " << mInverseIndexStorage->getIndex()->operator[](i)->load_factor() << std::endl;
            
//         }
    
}

neighborhood* InverseIndex::kneighbors(const umap_uniqueElement* pSignaturesMap, 
                                        const size_t pNneighborhood, const bool pDoubleElementsStorageCount) {
                                            // std::cout << "kneighbors inverseIndex" << std::endl;

    size_t doubleElements = 0;
    if (pDoubleElementsStorageCount) {
        doubleElements = mDoubleElementsStorageCount;
    } else {
        doubleElements = mDoubleElementsQueryCount;
    }
#ifdef OPENMP
    omp_set_dynamic(0);
#endif

    vvint* neighbors = new vvint();
    vvfloat* distances = new vvfloat();
    neighbors->resize(pSignaturesMap->size()+doubleElements);
    distances->resize(pSignaturesMap->size()+doubleElements);
    if (mChunkSize <= 0) {
        mChunkSize = ceil(mInverseIndexStorage->size() / static_cast<float>(mNumberOfCores));
    }
    
#ifdef OPENMP
#pragma omp parallel for schedule(static, mChunkSize) num_threads(mNumberOfCores)
#endif 

    for (size_t i = 0; i < pSignaturesMap->size(); ++i) {
        
        umap_uniqueElement::const_iterator instanceId = pSignaturesMap->begin();
        std::advance(instanceId, i);
        if (instanceId == pSignaturesMap->end()) continue;
        std::unordered_map<size_t, size_t> neighborhood;
        neighborhood.reserve(mMaxBinSize*2);
        const vsize_t* signature = instanceId->second.signature; 
        for (size_t j = 0; j < signature->size(); ++j) {
            size_t hashID = (*signature)[j];
            if (hashID != 0 && hashID != MAX_VALUE) {
                size_t collisionSize = 0; 
                
                const vsize_t* instances = mInverseIndexStorage->getElement(j, hashID);
                
                if (instances == NULL) continue;
                if (instances->size() != 0) {
                    collisionSize = instances->size();
                } else { 
                    continue;
                }
                
                if (collisionSize < mMaxBinSize && collisionSize > 0) {
                    for (size_t k = 0; k < instances->size(); ++k) {
                        neighborhood[(*instances)[k]] += 1;
                    }
                }
            }
        }
        
        if (neighborhood.size() == 0) {
            vint emptyVectorInt;
            emptyVectorInt.push_back(1);
            vfloat emptyVectorFloat;
            emptyVectorFloat.push_back(1);
#ifdef OPENMP
#pragma omp critical
#endif
            { // write vector to every instance with identical signatures

                for (size_t j = 0; j < instanceId->second.instances->size(); ++j) {
                    (*neighbors)[(*instanceId->second.instances)[j]] = emptyVectorInt;
                    (*distances)[(*instanceId->second.instances)[j]] = emptyVectorFloat;
                }
            }
            continue;
        }
         
        std::vector< sort_map > neighborhoodVectorForSorting;
        for (auto it = neighborhood.begin(); it != neighborhood.end(); ++it) {
            sort_map mapForSorting;
            mapForSorting.key = (*it).first;
            mapForSorting.val = (*it).second;
            neighborhoodVectorForSorting.push_back(mapForSorting);
        }
        size_t numberOfElementsToSort = pNneighborhood;
        if (pNneighborhood > neighborhoodVectorForSorting.size()) {
            numberOfElementsToSort = neighborhoodVectorForSorting.size();
        }
        
        std::partial_sort(neighborhoodVectorForSorting.begin(), 
                            neighborhoodVectorForSorting.begin()+numberOfElementsToSort, 
                            neighborhoodVectorForSorting.end(), mapSortDescByValue);
        size_t sizeOfNeighborhoodAdjusted;
        if (pNneighborhood == MAX_VALUE) {
            sizeOfNeighborhoodAdjusted = std::min(static_cast<size_t>(pNneighborhood), neighborhoodVectorForSorting.size());
        } else {
            sizeOfNeighborhoodAdjusted = std::min(static_cast<size_t>(pNneighborhood * mExcessFactor), neighborhoodVectorForSorting.size());
        }

        size_t count = 0;
        vvint neighborsForThisInstance(instanceId->second.instances->size());
        vvfloat distancesForThisInstance(instanceId->second.instances->size());

        for (size_t j = 0; j < neighborsForThisInstance.size(); ++j) {
            vint neighborhoodVector;
            std::vector<float> distanceVector;
            if (neighborhoodVectorForSorting[0].key != (*instanceId->second.instances)[j]) {
                neighborhoodVector.push_back((*instanceId->second.instances)[j]);
                distanceVector.push_back(0);
                ++count;
            }
            for (auto it = neighborhoodVectorForSorting.begin();
                    it != neighborhoodVectorForSorting.end(); ++it) {
                neighborhoodVector.push_back((*it).key);
                distanceVector.push_back(1 - ((*it).val / static_cast<float>(mMaximalNumberOfHashCollisions)));
                ++count;
                if (count >= sizeOfNeighborhoodAdjusted) {
                    neighborsForThisInstance[j] = neighborhoodVector;
                    distancesForThisInstance[j] = distanceVector;
                    break;
                }
            }
        }
        
#ifdef OPENMP
#pragma omp critical
#endif

        {   // write vector to every instance with identical signatures
       
            for (size_t j = 0; j < instanceId->second.instances->size(); ++j) {
                (*neighbors)[(*instanceId->second.instances)[j]] = neighborsForThisInstance[j];
                (*distances)[(*instanceId->second.instances)[j]] = distancesForThisInstance[j];
            }
        
        }
    }
    
    neighborhood* neighborhood_ = new neighborhood();
    neighborhood_->neighbors = neighbors;
    neighborhood_->distances = distances;
    return neighborhood_;
}