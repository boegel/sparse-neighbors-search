#include "hip/hip_runtime.h"
/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutors: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwigs-University Freiburg im Breisgau
**/

#include "inverseIndexCuda.h"
#include "kernel.h"

InverseIndexCuda::InverseIndexCuda(size_t pNumberOfHashFunctions, 
                                    size_t pShingle, size_t pShingleSize, 
                                    size_t pBlockSize) {
          mNumberOfHashFunctions = pNumberOfHashFunctions;
          mShingle = pShingle;
          mShingleSize = pShingleSize;
          mBlockSize = pBlockSize;
}
InverseIndexCuda::~InverseIndexCuda() {
    hipFree(mDev_FeatureList);
    hipFree(mDev_ValuesList);
    hipFree(mDev_SizeOfInstanceList);
}
void InverseIndexCuda::copyFittingDataToGpu(const SparseMatrixFloat* pRawData) {
    // memory for instances and their featureIds
    hipMalloc((void **) &mDev_FeatureList,
            pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t));
    // memory for the values of the features of the instances
    hipMalloc((void **) &mDev_ValuesList, 
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(float));
    // memory for the number of features per instance
    hipMalloc((void **) &mDev_SizeOfInstanceList,
            pRawData->getNumberOfInstances() * sizeof(size_t));
    
    // copy instances and their feature ids to the gpu
    hipMemcpy(mDev_FeatureList, pRawData->getSparseMatrixIndex(),
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t),
            hipMemcpyHostToDevice);
    // copy instances and their values for each feature to the gpu
    hipMemcpy(mDev_ValuesList, pRawData->getSparseMatrixValues(),
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(float),
            hipMemcpyHostToDevice);
    // copy the size of all instances to the gpu               
    hipMemcpy(mDev_SizeOfInstanceList, pRawData->getSparseMatrixSizeOfInstances(),
            pRawData->getNumberOfInstances() * sizeof(size_t),
            hipMemcpyHostToDevice);
}
void InverseIndexCuda::computeSignaturesFittingOnGpu(const SparseMatrixFloat* pRawData, 
                                                size_t pStartIndex, size_t pEndIndex, 
                                                size_t pNumberOfInstances, size_t pNumberOfBlocks, 
                                                size_t pNumberOfThreads, size_t pShingleFactor, 
                                                size_t pBlockSizeShingle,
                                                vvsize_t_p* pSignatures) {
    // check if enough memory is available on the gpu 
    size_t memory_total = 0;
    size_t memory_free = 0;
    size_t iterations = 1;
    size_t numberOfInstances = pEndIndex - pStartIndex;
    size_t signaturesSize = mNumberOfHashFunctions * pBlockSizeShingle / pShingleFactor;
    
    // memory for all signatures and memory for signatures on each block
    size_t neededMemory = numberOfInstances / iterations  * signaturesSize * sizeof(size_t);
    neededMemory += pNumberOfBlocks * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(size_t);
    hipMemGetInfo(&memory_free, &memory_total);
    // do i need more memory than it is free?
    if (neededMemory > memory_free) {
        iterations = ceil(neededMemory / static_cast<float>(memory_free));
    }
    
    size_t start = pStartIndex;
    size_t end = numberOfInstances / iterations;
  
    size_t windowSize = numberOfInstances / iterations;
    size_t* instancesHashValues = (size_t*) malloc(numberOfInstances / iterations * mNumberOfHashFunctions * sizeof(size_t));
    
    // memory for the inverse index on the gpu.
    // for each instance the number of hash functions
    hipMalloc((void **) &mDev_ComputedSignaturesPerInstance,
            numberOfInstances / iterations  * signaturesSize * sizeof(size_t));
    size_t* dev_SignaturesBlockSize;
    hipMalloc((void **) &dev_SignaturesBlockSize,
           pNumberOfBlocks * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(size_t));
     
    printf("start: %i, end: %i, iterations: %i\n", start, end, iterations);
    // compute the signatures on the gpu
    // do it in n iterations with equal sized chunks 
    // if the data would not fit on the ram of the gpu
    for (size_t i = 0; i < iterations; ++i) {
        // execute kernel on gpu
        fitCuda<<<pNumberOfBlocks, pNumberOfThreads>>>
        (mDev_FeatureList, 
        mDev_SizeOfInstanceList,  
        mNumberOfHashFunctions, 
        pRawData->getMaxNnz(),
                mDev_ComputedSignaturesPerInstance, 
                end, start, mBlockSize, mShingleSize, dev_SignaturesBlockSize);
        // copy results back to host      
        hipMemcpy(instancesHashValues, mDev_ComputedSignaturesPerInstance, 
                    numberOfInstances/iterations * signaturesSize * sizeof(size_t),
                    hipMemcpyDeviceToHost);
        // copy values into one vector per instance
        for(size_t i = start; i < end; ++i) {
            vsize_t* instance = new vsize_t(signaturesSize);
            for (size_t j = 0; j < signaturesSize; ++j) {
                (*instance)[j] = instancesHashValues[i*signaturesSize + j];
            }
            // printf("instance: %i\n", i);

            (*pSignatures)[i] = instance;
        }
        
        start = end+1;
        end = end + windowSize;
    }
    
    hipFree(mDev_ComputedSignaturesPerInstance);
    hipFree(dev_SignaturesBlockSize);
}
void InverseIndexCuda::computeSignaturesQueryOnGpu(const SparseMatrixFloat* pRawData, 
                                                size_t pStartIndex, size_t pEndIndex, 
                                                size_t pNumberOfInstances, size_t pNumberOfBlocks, 
                                                size_t pNumberOfThreads, size_t pShingleFactor, 
                                                size_t pBlockSizeShingle,
                                                vvsize_t_p* pSignatures) {
    size_t* dev_featureList;
    size_t* dev_sizeOfInstanceList;
    size_t* dev_computedSignaturesPerInstance;
    size_t numberOfInstances = pEndIndex - pStartIndex;
    size_t signaturesSize = mNumberOfHashFunctions * pBlockSizeShingle / pShingleFactor;
    
    size_t memory_total = 0;
    size_t memory_free = 0;
    size_t iterations = 1;
    // memory for all signatures and memory for signatures on each block
    // feature list memory
    size_t neededMemory = pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t);
    // memory for the number of features per instance
    neededMemory += pRawData->getNumberOfInstances() * sizeof(size_t);
    // memory for the signatures per instance
    neededMemory += numberOfInstances / iterations  * signaturesSize * sizeof(size_t);
    // memory for the signatures per instance for each block before shingle
    neededMemory += pNumberOfBlocks * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(size_t);
    hipMemGetInfo(&memory_free, &memory_total);
    // do i need more memory than it is free?
    if (neededMemory > memory_free) {
        iterations = ceil(neededMemory / static_cast<float>(memory_free));
    }
    // memory for instances and their featureIds
    hipMalloc((void **) &dev_featureList,
            pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t));
    // memory for the number of features per instance
    hipMalloc((void **) &dev_sizeOfInstanceList,
            pRawData->getNumberOfInstances() * sizeof(size_t));
    
    // copy instances and their feature ids to the gpu
    hipMemcpy(dev_featureList, pRawData->getSparseMatrixIndex(),
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(size_t),
            hipMemcpyHostToDevice);
    // copy the size of all instances to the gpu               
    hipMemcpy(dev_sizeOfInstanceList, pRawData->getSparseMatrixSizeOfInstances(),
            pRawData->getNumberOfInstances() * sizeof(size_t),
            hipMemcpyHostToDevice);
            
    
    size_t start = 0;
    size_t end = numberOfInstances / iterations;
    size_t windowSize = numberOfInstances / iterations;
    size_t* instancesHashValues = (size_t*) malloc(numberOfInstances / iterations * mNumberOfHashFunctions * sizeof(size_t));
    
    // size_t signaturesSize = mNumberOfHashFunctions * pBlockSizeShingle / pShingleFactor;
    // memory for the signatures on the gpu.
    // for each instance the number of hash functions
    hipMalloc((void **) &dev_computedSignaturesPerInstance,
            numberOfInstances / iterations  * signaturesSize * sizeof(size_t));
    size_t* dev_signaturesBlockSize;
    hipMalloc((void **) &dev_signaturesBlockSize,
           pNumberOfBlocks * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(size_t));
    // compute the signatures on the gpu
    // do it in n iterations with equal sized chunks 
    // if the data would not fit on the ram of the gpu
    for (size_t i = 0; i < iterations; ++i) {
        // execute kernel on gpu
        fitCuda<<<pNumberOfBlocks, pNumberOfThreads>>>
        (dev_featureList, 
        dev_sizeOfInstanceList,  
        mNumberOfHashFunctions, 
        pRawData->getMaxNnz(),
                dev_computedSignaturesPerInstance, 
                end, start, mBlockSize, mShingleSize, dev_signaturesBlockSize);
        // copy results back to host      
        hipMemcpy(instancesHashValues, dev_computedSignaturesPerInstance, 
                    numberOfInstances/iterations * signaturesSize * sizeof(size_t),
                    hipMemcpyDeviceToHost);
        // copy values into one vector per instance
        for(size_t i = start; i < end; ++i) {
            vsize_t* instance = new vsize_t(signaturesSize);
            for (size_t j = 0; j < signaturesSize; ++j) {
                (*instance)[j] = instancesHashValues[i*signaturesSize + j];
            }
            (*pSignatures)[i] = instance;
        }
        
        start = end+1;
        end = end + windowSize;
    }
    
    hipFree(dev_computedSignaturesPerInstance);
    hipFree(dev_signaturesBlockSize);
    hipFree(dev_featureList);       
    hipFree(dev_computedSignaturesPerInstance);       
           
}
void InverseIndexCuda::computeHitsOnGpu(std::vector<vvsize_t_p*>* pHitsPerInstance, 
                                                neighborhood* pNeighborhood, 
                                                size_t pNeighborhoodSize,
                                                size_t pNumberOfInstances,
                                                const size_t pNumberOfBlocksHistogram,
                                                const size_t pNumberOfThreadsHistogram,
                                                const size_t pNumberOfBlocksDistance,
                                                const size_t pNumberOfThreadsDistance,
                                                size_t pFast, size_t pDistance,
                                                size_t pExcessFactor, size_t pMaxNnz) {
                                                    
//     struct hits {
//     size_t* instances;
//     size_t size;
// };
    // size_t* hitsPerInstance_realloc = NULL;
    //  = (size_t*) malloc(pHitsPerInstance->)
    size_t* elementsPerInstance = (size_t*) malloc(pHitsPerInstance->size() * sizeof(size_t));
    size_t counter = 0;
    
    hits* data = (hits*) malloc(sizeof(hits) * pHitsPerInstance->size());
    hits* dev_data_inner = (hits*) malloc(sizeof(hits) * pHitsPerInstance->size());
    // size_t i = 0;
    // return;
    // std::cout << "Size of pHits: " << pHitsPerInstance->size() << " pNumberofInstances: " << pNumberOfInstances;
    // std::cout << std::endl;
    size_t counterHitsPerInstance = 0;
    for (size_t i = 0; i < pHitsPerInstance->size(); ++i) {
        for (auto itQueryInstance = (*pHitsPerInstance)[i]->begin(); 
                itQueryInstance != (*pHitsPerInstance)[i]->end(); ++itQueryInstance) {
            for(auto itInstance = (*itQueryInstance)->begin(); 
                itInstance != (*itQueryInstance)->end(); ++itInstance) {
                    // hitsPerInstance[counterHitsPerInstance] = *itInstance;
                    ++counter;
            }
        }
        size_t* instances = (size_t*) malloc(sizeof(size_t) * counter);
        size_t j = 0;
        for (auto itQueryInstance = (*pHitsPerInstance)[i]->begin(); 
                itQueryInstance != (*pHitsPerInstance)[i]->end(); ++itQueryInstance) {
            for(auto itInstance = (*itQueryInstance)->begin(); 
                itInstance != (*itQueryInstance)->end(); ++itInstance) {
                    instances[j] = *itInstance;
                    ++j;
            }
        }
        data[i].instances = instances;
        data[i].size = counter;
        counter = 0;
    }
    memcpy(dev_data_inner, data, sizeof(hits) * pHitsPerInstance->size()));
    
    for (size_t i = 0; i < pHitsPerInstance->size(); ++i) {
        hipMalloc(&(dev_data[i].instances), data[i].size*sizeof(size_t));
        hipMemcpy(dev_data[i].instances, data[i].instances, data[i].size*sizeof(size_t));
    }
    hits* dev_data;
    hipMalloc(&dev_data, sizeof(hits) * pHitsPerInstance->size());
    hipMemcpy(dev_data, dev_data_inner, sizeof(hits) * pHitsPerInstance->size());
    
    // histogram* pHistogram, radixSortingMemory* pRadixSortMemory,
                                            // sortedHistogram* pHistogramSorted, 
    // reserve space for histogram on device
    histogram* dev_histogram_inner = (histogram*) malloc(pNumberOfBlocksHistogram*sizeof(histogram));
    for (size_t i = 0; i < pNumberOfBlocksHistogram; ++i) {
        hipMalloc(&(dev_histogram_inner[i].instances), pNumberOfInstances * sizeof(size_t));
    }
    histogram* dev_histogram;
    hipMalloc(&dev_histogram, pNumberOfBlocksHistogram*sizeof(histogram));
    hipMemcpy(dev_histogram, dev_histogram_inner, pNumberOfBlocksHistogram*sizeof(histogram));
    
    
    // reserve for sorting on device
    radixSortingMemory* dev_radixSortingMemory_inner = (radixSortingMemory*) malloc(pNumberOfBlocksHistogram*sizeof(radixSortingMemory));
    for (size_t i = 0; i < pNumberOfBlocksHistogram; ++i) {
        hipMalloc(&(dev_radixSortingMemory_inner[i].bucketNull), pNumberOfInstances * sizeof(int2));
        hipMalloc(&(dev_radixSortingMemory_inner[i].bucketOne), pNumberOfInstances * sizeof(int2));
    }
    
    radixSortingMemory* dev_radixSortingMemory;
    hipMalloc(&dev_radixSortingMemory, pNumberOfBlocksHistogram*sizeof(radixSortingMemory));
    hipMemcpy(dev_radixSortingMemory, dev_radixSortingMemory_inner, pNumberOfBlocksHistogram*sizeof(radixSortingMemory));
    
    // space for sorted histogram
    sortedHistogram* dev_sortedHistogram_inner = (sortedHistogram*) malloc(pNumberOfBlocksHistogram*sizeof(sortedHistogram));
    for (size_t i = 0; i < pNumberOfBlocksHistogram; ++i) {
        hipMalloc(&(dev_sortedHistogram_inner[i].instances), pNumberOfInstances * sizeof(int2));
    }
    
    sortedHistogram* dev_sortedHistogram;
    hipMalloc(&dev_sortedHistogram, pNumberOfBlocksHistogram*sizeof(sortedHistogram));
    hipMemcpy(dev_sortedHistogram, dev_sortedHistogram_inner, pNumberOfBlocksHistogram*sizeof(sortedHistogram));
    
    
    // printf("foo"); 
    size_t memory_free;
    size_t memory_total;
    size_t iterations = 1;
    vsize_t instancesPerIteration = {pNumberOfInstances};
    // size_t 
    // memory for all instances and their hits
    size_t needed_memory = counterHitsPerInstance * sizeof(size_t);
    // memory for the histogram
    needed_memory += pNumberOfBlocksHistogram*pNumberOfInstances * sizeof(int);
    // memory for number of elements per instance
    needed_memory += pNumberOfInstances * sizeof(size_t);
    // memory for radix sort
    needed_memory += pNumberOfBlocksHistogram * pNumberOfInstances * 2 * 2 * sizeof(int);
    // memory for sorted instances
    needed_memory += pNumberOfBlocksHistogram * pNumberOfInstances * 2 * sizeof(int);
    // get memory usage from gpu
    hipMemGetInfo(&memory_free, &memory_total);
    // enough memory on the gpu plus an buffer of 1MB
    // if (memory_free >= needed_memory+1024*8*1024) {
    //     iterations = ceil(needed_memory / static_cast<float>(memory_free));
    //     if (iterations > elementsPerInstance.size()) {
    //         printf("Sorry your dataset is too big to be computed on your GPU. Please use CPU only mode");
    //     }
    //     size_t elementsPerIteration = elementsPerInstance.size() / iteration;
    //     for (size_t i = 0; i < iterations; ++i) {
    //         size_t numberOfElements = 0;
    //         for (size_t j = elementsPerIteration * i; j < elementsPerIteration ; ++j) {
    //               numberOfElements += elementsPerIteration[j];
    //         }
    //         instancesPerIteration.push_back(numberOfElements);
    //     }
    // }
    
    size_t* dev_Neighborhood;
    float* dev_Distances;
    size_t* neighborhood = (size_t*) malloc(pHitsPerInstance->size() * pNeighborhoodSize * sizeof(size_t));
    float* distances = (float*) malloc(pHitsPerInstance->size() * pNeighborhoodSize * sizeof(float));
    int2* histogram = (int2*) malloc(pNumberOfBlocksHistogram * pNumberOfInstances * sizeof(int2));
    // size_t* instancesHashValues = (size_t*) malloc(pRawData->getNumberOfInstances() / iterations * mNumberOfHashFunctions * sizeof(size_t));
    
    hipMalloc((void **) &dev_Neighborhood,
                pHitsPerInstance->size() * pNeighborhoodSize * pExcessFactor * sizeof(size_t));
    hipMalloc((void **) &dev_Distances,
                pHitsPerInstance->size() * pNeighborhoodSize * sizeof(float));
    vvint* neighborsVector = new vvint(pHitsPerInstance->size());
    vvfloat* distancesVector = new vvfloat(pHitsPerInstance->size());
    for (size_t i = 0; i < iterations; ++i) {
        createSortedHistogramsCuda<<<pNumberOfBlocksHistogram, pNumberOfThreadsHistogram>>>
                    (dev_data, pNumberOfInstances, dev_histogram,
                    dev_radixSortingMemory, dev_sortedHistogram, 
                    dev_NumberOfPossibleNeighbors, 
                    pNeighborhoodSize, pExcessFactor,
                    dev_Neighborhood, dev_Distances, pFast);
        hipMemcpy(histogram, dev_HistogramSorted, pNumberOfBlocksHistogram * pNumberOfInstances *sizeof(int2)
                        ,hipMemcpyDeviceToHost); 
        printf("pNumberOfInstances: %i", pNumberOfInstances);
        printf("\n\nunsorted histogram: ");
        for (size_t j = 0; j <  pNumberOfInstances; j += 1) {
                printf("id: %i, count: %i", histogram[j].y, histogram[j].x);
        }
        printf("\n\n");
        if (!pFast) {
            if (pDistance) {
                euclideanDistanceCuda<<<pNumberOfBlocksDistance, pNumberOfThreadsDistance>>>
                                        (dev_sortedHistogram, dev_NumberOfPossibleNeighbors, 
                                        pNumberOfInstances,
                                        mDev_FeatureList, mDev_ValuesList,
                                        mDev_SizeOfInstanceList, pMaxNnz,
                                        dev_RadixSortMemory, pNeighborhoodSize,
                                        dev_Neighborhood, dev_Distances);
                
            } else {
                cosineSimilarityCuda<<<pNumberOfBlocksDistance, pNumberOfThreadsDistance>>>
                                        (dev_HistogramSorted, dev_NumberOfPossibleNeighbors, 
                                        rangeBetweenInstances, pNumberOfInstances,
                                        mDev_FeatureList, mDev_ValuesList,
                                        mDev_SizeOfInstanceList, pMaxNnz,
                                        dev_RadixSortMemory, pNeighborhoodSize,
                                        dev_Neighborhood, dev_Distances);
            }
        }
        
        hipMemcpy(neighborhood, dev_Neighborhood,
                    pHitsPerInstance->size() * pNeighborhoodSize * sizeof(int),
                    hipMemcpyDeviceToHost);
        hipMemcpy(distances, dev_Distances,
                    pHitsPerInstance->size() * pNeighborhoodSize * sizeof(float),
                    hipMemcpyDeviceToHost);
                    
        hipFree(dev_Neighborhood);
        hipFree(dev_Distances);
        // transfer to neighorhood layout
       
        
        for (size_t i = 0; i < pHitsPerInstance->size(); ++i) {
            vint neighbors_;
            vfloat distances_;
            for (size_t j = 0; j < pNeighborhoodSize; ++j) {
                neighbors_.push_back(neighborhood[i*pNeighborhoodSize + j]);
                distances_.push_back(distances[i*pNeighborhoodSize + j]);
            }
            (*neighborsVector)[i] = neighbors_;
            (*distancesVector)[i] = distances_;
        }
    }
     
    pNeighborhood->neighbors = neighborsVector;
    pNeighborhood->distances = distancesVector;
    free(neighborhood);
    free(distances);
    free(elementsPerInstance);
    hipFree(dev_HitsPerInstances);
    hipFree(dev_ElementsPerInstances);
    hipFree(dev_Histogram);
    hipFree(dev_HistogramSorted);
    hipFree(dev_RadixSortMemory);
    hipFree(dev_NumberOfPossibleNeighbors);
     // return it
     // delete memory
     // 
     // check if everything is fitting in gpu memory,
     // loop if not.              
}
