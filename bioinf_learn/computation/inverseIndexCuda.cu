#include "hip/hip_runtime.h"
/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutors: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwigs-University Freiburg im Breisgau
**/

#include "inverseIndexCuda.h"
#include "kernel.h"

InverseIndexCuda::InverseIndexCuda(size_t pNumberOfHashFunctions, 
                                    size_t pShingle, size_t pShingleSize, 
                                    size_t pBlockSize, size_t pHashAlgorithm) {
    mNumberOfHashFunctions = pNumberOfHashFunctions;
    mShingle = pShingle;
    mShingleSize = pShingleSize;
    mBlockSize = pBlockSize;
    mHashAlgorithm = pHashAlgorithm;
}
InverseIndexCuda::~InverseIndexCuda() {
    hipFree(mDev_FeatureList);
    hipFree(mDev_ValuesList);
    hipFree(mDev_SizeOfInstanceList);
    hipFree(mDev_JumpLength);
    hipFree(mDev_DotProduct);
}
void InverseIndexCuda::copyDataToGpu(SparseMatrixFloat* pRawData, int** pDevFeatureList,
                                      float** pDevValueList, size_t** pSizeList) {

    // memory for the number of features per instance
    hipMalloc((void **) &(*pSizeList),
           sizeof(size_t) * pRawData->size());
    // copy the size of all instances to the gpu               
    hipMemcpy((*pSizeList), pRawData->getSparseMatrixSizeOfInstances(),
            sizeof(size_t) * pRawData->size(),
            hipMemcpyHostToDevice);
    
    // memory for instances and their featureIds
    hipMalloc((void **) &(*pDevFeatureList),
            pRawData->size() * pRawData->getMaxNnz() * sizeof(int));
    // memory for the values of the features of the instances
    hipMalloc((void **) &(*pDevValueList), 
                pRawData->size() * pRawData->getMaxNnz() * sizeof(float));
    
    // copy instances and their feature ids to the gpu
    hipMemcpy((*pDevFeatureList), pRawData->getSparseMatrixIndex(),
                pRawData->size() * pRawData->getMaxNnz() * sizeof(int),
            hipMemcpyHostToDevice);
    
    hipMemcpy((*pDevValueList), pRawData->getSparseMatrixValues(),
                pRawData->size() * pRawData->getMaxNnz() * sizeof(float),
            hipMemcpyHostToDevice);
    // printf("mDev_FeatureListCOPY, %u\n", (*pDevFeatureList));
    // printf("&mDev_FeatureListCOPY, %u\n", &(*pDevFeatureList));
    // printf("mDev_FeatureListCOPY, %u\n", (*pDevValueList));
    // printf("&mDev_FeatureListCOPY, %u\n", &(*pDevValueList));
    // printf("mDev_FeatureListCOPY, %u\n", (*pSizeList));
    // printf("&mDev_FeatureListCOPY, %u\n", &(*pSizeList));
}
void InverseIndexCuda::computeSignaturesFittingOnGpu(SparseMatrixFloat* pRawData, 
                                                size_t pStartIndex, size_t pEndIndex, 
                                                size_t pNumberOfInstances, size_t pNumberOfBlocks, 
                                                size_t pNumberOfThreads, size_t pShingleFactor, 
                                                size_t pBlockSizeShingle,
                                                vvsize_t_p* pSignatures, size_t pRangeK) {
    // copy data to gpu
    // printf("mDev_FeatureList, %u\n", mDev_FeatureList);
    // printf("&mDev_FeatureList, %u\n", &mDev_FeatureList);
    // printf("mDev_ValuesList, %u\n", mDev_ValuesList);
    // printf("&mDev_ValuesList, %u\n", &mDev_ValuesList);
    // printf("mDev_SizeOfInstanceList, %u\n", mDev_SizeOfInstanceList);
    // printf("&mDev_SizeOfInstanceList, %u\n", &mDev_SizeOfInstanceList);
    // printf("%i\n", __LINE__);
    copyDataToGpu(pRawData, &mDev_FeatureList, &mDev_ValuesList, &mDev_SizeOfInstanceList);  
    // printf("%i\n", __LINE__);
                                               
    size_t signaturesSize = ceil(mNumberOfHashFunctions * pBlockSizeShingle / (float) pShingleFactor);
    // printf("%i\n", __LINE__);
   
    int* instancesHashValues = (int*) malloc(pRawData->size() * signaturesSize * sizeof(int));
    // printf("%i\n", __LINE__);
    
    // memory for the inverse index on the gpu.
    // for each instance the number of hash functions
    hipMalloc((void **) &mDev_ComputedSignaturesPerInstance,
            pRawData->size() * signaturesSize * sizeof(int));
    int* dev_SignaturesBlockSize;
    hipMalloc((void **) &dev_SignaturesBlockSize,
           128 * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(int));
    // printf("%i\n", __LINE__);
     
     
    // cuda memory for dot products dot<X, X>
    hipMalloc((void **) &mDev_DotProduct, sizeof(float) * pRawData->size());
  
    // printf("%i\n", __LINE__);
 
    
        // execute kernel on gpu
        if (mHashAlgorithm == 0) {
    // printf("%i\n", __LINE__);
            
            fitCudaMinHash<<<128, 128>>>
            (mDev_FeatureList, 
            mDev_SizeOfInstanceList,  
            mNumberOfHashFunctions, 
            pRawData->getMaxNnz(),
            mDev_ComputedSignaturesPerInstance, 
            pRawData->size(), 0, mBlockSize, mShingleSize, dev_SignaturesBlockSize);
            hipDeviceSynchronize();
    // printf("%i\n", __LINE__);
            
        } else {
            // fitCudaWtaHash<<<128, 128>>>
            // (mDev_FeatureList, 
            // mDev_SizeOfInstanceList,  
            // mNumberOfHashFunctions, 
            // mDev_JumpLength,
            //         mDev_ComputedSignaturesPerInstance, 
            //         end, start, mBlockSize, mShingleSize, dev_SignaturesBlockSize);
        }
        // dotProductSingle<<<128, 128>>>(mDev_FeatureList, mDev_ValuesList, mDev_SizeOfInstanceList,
        //                                 mDev_JumpLength, pRawData->size(), mDev_DotProduct);
        //     hipDeviceSynchronize();
    // printf("%i\n", __LINE__);
                                        
        // copy results back to host  
        hipMemcpy(instancesHashValues, mDev_ComputedSignaturesPerInstance, 
                    pRawData->size() * signaturesSize * sizeof(int),
                    hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    // printf("%i\n", __LINE__);
                    
        // copy values into one vector per instance
        for(size_t i = 0; i < pRawData->size(); ++i) {
            vsize_t* instance = new vsize_t(signaturesSize);
            for (size_t j = 0; j < signaturesSize; ++j) {
                (*instance)[j] = static_cast<size_t> (instancesHashValues[i*signaturesSize + j]);
            }
            (*pSignatures)[i] = instance;
        }
    // printf("%i\n", __LINE__);
    
    hipFree(mDev_ComputedSignaturesPerInstance);
    hipFree(dev_SignaturesBlockSize);
    dotProductSingle<<<128, 128>>>(mDev_FeatureList, mDev_ValuesList, mDev_SizeOfInstanceList,
                                        pRawData->size(), pRawData->getMaxNnz(), mDev_DotProduct);
    hipDeviceSynchronize();
    printf("DotproductInverse: %u\n", mDev_DotProduct);
}
void InverseIndexCuda::computeSignaturesQueryOnGpu(SparseMatrixFloat* pRawData, 
                                                size_t pStartIndex, size_t pEndIndex, 
                                                size_t pNumberOfInstances, size_t pNumberOfBlocks, 
                                                size_t pNumberOfThreads, size_t pShingleFactor, 
                                                size_t pBlockSizeShingle,
                                                vvsize_t_p* pSignatures, size_t pRangeK) {
                                                    
//    // memory for the number of features per instance
//     hipMalloc((void **) &(*pSizeList),
//            sizeof(size_t) * pRawData->size());
//     // copy the size of all instances to the gpu               
//     hipMemcpy((*pSizeList), pRawData->getSparseMatrixSizeOfInstances(),
//             sizeof(size_t) * pRawData->size(),
//             hipMemcpyHostToDevice);
    
//     // memory for instances and their featureIds
//     hipMalloc((void **) &(*pDevFeatureList),
//             pRawData->size() * pRawData->getMaxNnz() * sizeof(int));
//     // memory for the values of the features of the instances
//     hipMalloc((void **) &(*pDevValueList), 
//                 pRawData->size() * pRawData->getMaxNnz() * sizeof(float));
    
//     // copy instances and their feature ids to the gpu
//     hipMemcpy((*pDevFeatureList), pRawData->getSparseMatrixIndex(),
//                 pRawData->size() * pRawData->getMaxNnz() * sizeof(int),
//             hipMemcpyHostToDevice);
    
//     hipMemcpy((*pDevValueList), pRawData->getSparseMatrixValues(),
//                 pRawData->size() * pRawData->getMaxNnz() * sizeof(float),
//             hipMemcpyHostToDevice);
                                                           
//    size_t signaturesSize = ceil(mNumberOfHashFunctions * pBlockSizeShingle / (float) pShingleFactor);
   
//     size_t* instancesHashValues = (size_t*) malloc(pRawData->size() * signaturesSize * sizeof(size_t));
    
//     // memory for the inverse index on the gpu.
//     // for each instance the number of hash functions
//     hipMalloc((void **) &mDev_ComputedSignaturesPerInstance,
//             pRawData->size() * signaturesSize * sizeof(int));
//     int* dev_SignaturesBlockSize;
//     hipMalloc((void **) &dev_SignaturesBlockSize,
//            128 * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(int));
     
     
//     // cuda memory for dot products dot<X, X>
//     hipMalloc((void **) &mDev_DotProduct, sizeof(float) * pRawData->size());
  
 
    
//         // execute kernel on gpu
//         if (mHashAlgorithm == 0) {
//             fitCudaMinHash<<<128, 128>>>
//             (mDev_FeatureList, 
//             mDev_SizeOfInstanceList,  
//             mNumberOfHashFunctions, 
//             pRawData->getMaxNnz(),
//             mDev_ComputedSignaturesPerInstance, 
//             pRawData->size(), 0, mBlockSize, mShingleSize, dev_SignaturesBlockSize);
//             hipDeviceSynchronize();
//         } else {
//             // fitCudaWtaHash<<<128, 128>>>
//             // (mDev_FeatureList, 
//             // mDev_SizeOfInstanceList,  
//             // mNumberOfHashFunctions, 
//             // mDev_JumpLength,
//             //         mDev_ComputedSignaturesPerInstance, 
//             //         end, start, mBlockSize, mShingleSize, dev_SignaturesBlockSize);
//         }
//         // dotProductSingle<<<128, 128>>>(mDev_FeatureList, mDev_ValuesList, mDev_SizeOfInstanceList,
//         //                                 mDev_JumpLength, pRawData->size(), mDev_DotProduct);
//         //     hipDeviceSynchronize();
                                        
//         // copy results back to host  
//         hipMemcpy(instancesHashValues, mDev_ComputedSignaturesPerInstance, 
//                     pRawData->size() * signaturesSize * sizeof(int),
//                     hipMemcpyDeviceToHost);
//         hipDeviceSynchronize();
                    
//         // copy values into one vector per instance
//         for(size_t i = 0; i < pRawData->size(); ++i) {
//             vsize_t* instance = new vsize_t(signaturesSize);
//             for (size_t j = 0; j < signaturesSize; ++j) {
//                 (*instance)[j] = static_cast<size_t> (instancesHashValues[i*signaturesSize + j]);
//             }
//             (*pSignatures)[i] = instance;
//         }
    
//     hipFree(mDev_ComputedSignaturesPerInstance);
//     hipFree(dev_SignaturesBlockSize);
//     // dotProductSingle<<<128, 128>>>(mDev_FeatureList, mDev_ValuesList, mDev_SizeOfInstanceList,
//     //                                     pRawData->size(), pRawData->getMaxNnz(), mDev_DotProduct);
//     hipDeviceSynchronize();
}