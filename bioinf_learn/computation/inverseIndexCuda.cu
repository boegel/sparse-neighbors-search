#include "hip/hip_runtime.h"
/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutors: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwigs-University Freiburg im Breisgau
**/

#include "inverseIndexCuda.h"
#include "kernel.h"

InverseIndexCuda::InverseIndexCuda(size_t pNumberOfHashFunctions, 
                                    size_t pShingle, size_t pShingleSize, 
                                    size_t pBlockSize) {
          mNumberOfHashFunctions = pNumberOfHashFunctions;
          mShingle = pShingle;
          mShingleSize = pShingleSize;
          mBlockSize = pBlockSize;
}
InverseIndexCuda::~InverseIndexCuda() {
    hipFree(mDev_FeatureList);
    hipFree(mDev_ValuesList);
    hipFree(mDev_SizeOfInstanceList);
}
void InverseIndexCuda::copyFittingDataToGpu(const SparseMatrixFloat* pRawData) {
    // memory for instances and their featureIds
    hipMalloc((void **) &mDev_FeatureList,
            pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(int));
    // memory for the values of the features of the instances
    hipMalloc((void **) &mDev_ValuesList, 
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(float));
    // memory for the number of features per instance
    hipMalloc((void **) &mDev_SizeOfInstanceList,
            pRawData->getNumberOfInstances() * sizeof(int));
    
    int* dev_index = (int*) malloc(sizeof(int) * pRawData->getMaxNnz() * pRawData->getNumberOfInstances());
    for (unsigned int i = 0; i < pRawData->getMaxNnz() * pRawData->getNumberOfInstances(); ++i) {
        dev_index[i] = static_cast<int>(pRawData->getSparseMatrixIndex()[i]);
    }
    // copy instances and their feature ids to the gpu
    hipMemcpy(mDev_FeatureList, dev_index,
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(int),
            hipMemcpyHostToDevice);
    // copy instances and their values for each feature to the gpu
    float* dev_values = (float*) malloc(sizeof(float) * pRawData->getMaxNnz() * pRawData->getNumberOfInstances());
    for (unsigned int i = 0; i < pRawData->getMaxNnz() * pRawData->getNumberOfInstances(); ++i) {
        dev_values[i] = static_cast<float>(pRawData->getSparseMatrixValues()[i]);
    }
    hipMemcpy(mDev_ValuesList, dev_values,
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(float),
            hipMemcpyHostToDevice);
            
    int* dev_sizes = (int*) malloc(sizeof(int) * pRawData->getNumberOfInstances());
    for (unsigned int i = 0; i < pRawData->getNumberOfInstances(); ++i) {
        dev_sizes[i] = static_cast<int>(pRawData->getSparseMatrixSizeOfInstances()[i]);
    }
    // copy the size of all instances to the gpu               
    hipMemcpy(mDev_SizeOfInstanceList, dev_sizes,
            pRawData->getNumberOfInstances() * sizeof(int),
            hipMemcpyHostToDevice);
     for (unsigned int i = 0; i < pRawData->getNumberOfInstances(); ++i) {
        // printf ("instanceId: %i, size: %i\n", i, dev_sizes[i]);
     }
     
     
    //   for (int i = 0; i < pRawData->getMaxNnz(); ++i) {
    //         // if (i % 100 == 0) {
    //             // for (int j = 0; j < pSizeOfCandidates[i]; ++j) {
    //                 // if (j % 20 == 0) {
    //                     printf ("feature: %i, value: %f\n", dev_index[i], pRawData->getSparseMatrixValues()[i]);
                        
    //                 // }
    //             // }
    //         // }   
    //     }
    free(dev_index);
        free(dev_values);
     free(dev_sizes);
}
void InverseIndexCuda::computeSignaturesFittingOnGpu(const SparseMatrixFloat* pRawData, 
                                                size_t pStartIndex, size_t pEndIndex, 
                                                size_t pNumberOfInstances, size_t pNumberOfBlocks, 
                                                size_t pNumberOfThreads, size_t pShingleFactor, 
                                                size_t pBlockSizeShingle,
                                                vvsize_t_p* pSignatures) {
    // check if enough memory is available on the gpu 
    size_t memory_total = 0;
    size_t memory_free = 0;
    size_t iterations = 1;
    size_t numberOfInstances = pEndIndex - pStartIndex;
    size_t signaturesSize = mNumberOfHashFunctions * pBlockSizeShingle / pShingleFactor;
    
    // memory for all signatures and memory for signatures on each block
    size_t neededMemory = numberOfInstances / iterations  * signaturesSize * sizeof(int);
    neededMemory += pNumberOfBlocks * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(int);
    hipMemGetInfo(&memory_free, &memory_total);
    // do i need more memory than it is free?
    if (neededMemory > memory_free) {
        iterations = ceil(neededMemory / static_cast<float>(memory_free));
    }
    
    size_t start = pStartIndex;
    size_t end = numberOfInstances / iterations;
  
    size_t windowSize = numberOfInstances / iterations;
    int* instancesHashValues = (int*) malloc(numberOfInstances / iterations * mNumberOfHashFunctions * sizeof(int));
    
    // memory for the inverse index on the gpu.
    // for each instance the number of hash functions
    hipMalloc((void **) &mDev_ComputedSignaturesPerInstance,
            numberOfInstances / iterations  * signaturesSize * sizeof(int));
    int* dev_SignaturesBlockSize;
    hipMalloc((void **) &dev_SignaturesBlockSize,
           pNumberOfBlocks * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(int));
     
    // printf("start: %i, end: %i, iterations: %i\n", start, end, iterations);
    // compute the signatures on the gpu
    // do it in n iterations with equal sized chunks 
    // if the data would not fit on the ram of the gpu
    for (size_t i = 0; i < iterations; ++i) {
        // execute kernel on gpu
        fitCuda<<<pNumberOfBlocks, pNumberOfThreads>>>
        (mDev_FeatureList, 
        mDev_SizeOfInstanceList,  
        mNumberOfHashFunctions, 
        pRawData->getMaxNnz(),
                mDev_ComputedSignaturesPerInstance, 
                end, start, mBlockSize, mShingleSize, dev_SignaturesBlockSize);
        // copy results back to host      
        hipMemcpy(instancesHashValues, mDev_ComputedSignaturesPerInstance, 
                    numberOfInstances/iterations * signaturesSize * sizeof(int),
                    hipMemcpyDeviceToHost);
        // copy values into one vector per instance
        for(size_t i = start; i < end; ++i) {
            vsize_t* instance = new vsize_t(signaturesSize);
            for (size_t j = 0; j < signaturesSize; ++j) {
                (*instance)[j] = static_cast<size_t>(instancesHashValues[i*signaturesSize + j]);
            }
            // printf("instance: %i\n", i);

            (*pSignatures)[i] = instance;
        }
        
        start = end+1;
        end = end + windowSize;
    }
    
    hipFree(mDev_ComputedSignaturesPerInstance);
    hipFree(dev_SignaturesBlockSize);
}
void InverseIndexCuda::computeSignaturesQueryOnGpu(const SparseMatrixFloat* pRawData, 
                                                size_t pStartIndex, size_t pEndIndex, 
                                                size_t pNumberOfInstances, size_t pNumberOfBlocks, 
                                                size_t pNumberOfThreads, size_t pShingleFactor, 
                                                size_t pBlockSizeShingle,
                                                vvsize_t_p* pSignatures) {
    int* dev_featureList;
    int* dev_sizeOfInstanceList;
    int* dev_computedSignaturesPerInstance;
    int numberOfInstances = pEndIndex - pStartIndex;
    int signaturesSize = mNumberOfHashFunctions * pBlockSizeShingle / pShingleFactor;
    
    size_t memory_total = 0;
    size_t memory_free = 0;
    int iterations = 1;
    // memory for all signatures and memory for signatures on each block
    // feature list memory
    int neededMemory = pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(int);
    // memory for the number of features per instance
    neededMemory += pRawData->getNumberOfInstances() * sizeof(int);
    // memory for the signatures per instance
    neededMemory += numberOfInstances / iterations  * signaturesSize * sizeof(int);
    // memory for the signatures per instance for each block before shingle
    neededMemory += pNumberOfBlocks * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(int);
    hipMemGetInfo(&memory_free, &memory_total);
    // do i need more memory than it is free?
    if (neededMemory > memory_free) {
        iterations = ceil(neededMemory / static_cast<float>(memory_free));
    }
    // memory for instances and their featureIds
    hipMalloc((void **) &dev_featureList,
            pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(int));
    // memory for the number of features per instance
    hipMalloc((void **) &dev_sizeOfInstanceList,
            pRawData->getNumberOfInstances() * sizeof(int));
    
    // copy instances and their feature ids to the gpu
    int* dev_index = (int*) malloc(sizeof(int) * pRawData->getMaxNnz() * pRawData->getNumberOfInstances());
    for (unsigned int i = 0; i < pRawData->getMaxNnz() * pRawData->getNumberOfInstances(); ++i) {
        dev_index[i] = static_cast<int>(pRawData->getSparseMatrixIndex()[i]);
    }
    hipMemcpy(dev_featureList, dev_index,
                pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(int),
            hipMemcpyHostToDevice);
    free(dev_index);
    // copy the size of all instances to the gpu    
    int* dev_sizes = (int*) malloc(sizeof(int) * pRawData->getNumberOfInstances());
    for (unsigned int i = 0; i < pRawData->getNumberOfInstances(); ++i) {
        dev_sizes[i] = static_cast<int>(pRawData->getSparseMatrixSizeOfInstances()[i]);
    }           
    hipMemcpy(dev_sizeOfInstanceList, dev_sizes,
            pRawData->getNumberOfInstances() * sizeof(int),
            hipMemcpyHostToDevice);
     free(dev_sizes);       
    
    int start = 0;
    int end = numberOfInstances / iterations;
    int windowSize = numberOfInstances / iterations;
    int* instancesHashValues = (int*) malloc(numberOfInstances / iterations * mNumberOfHashFunctions * sizeof(int));
    
    // size_t signaturesSize = mNumberOfHashFunctions * pBlockSizeShingle / pShingleFactor;
    // memory for the signatures on the gpu.
    // for each instance the number of hash functions
    hipMalloc((void **) &dev_computedSignaturesPerInstance,
            numberOfInstances / iterations  * signaturesSize * sizeof(int));
    int* dev_signaturesBlockSize;
    hipMalloc((void **) &dev_signaturesBlockSize,
           pNumberOfBlocks * mNumberOfHashFunctions * pBlockSizeShingle * sizeof(int));
    // compute the signatures on the gpu
    // do it in n iterations with equal sized chunks 
    // if the data would not fit on the ram of the gpu
    for (size_t i = 0; i < iterations; ++i) {
        // execute kernel on gpu
        fitCuda<<<pNumberOfBlocks, pNumberOfThreads>>>
        (dev_featureList, 
        dev_sizeOfInstanceList,  
        mNumberOfHashFunctions, 
        pRawData->getMaxNnz(),
                dev_computedSignaturesPerInstance, 
                end, start, mBlockSize, mShingleSize, dev_signaturesBlockSize);
        // copy results back to host      
        hipMemcpy(instancesHashValues, dev_computedSignaturesPerInstance, 
                    numberOfInstances/iterations * signaturesSize * sizeof(int),
                    hipMemcpyDeviceToHost);
        // copy values into one vector per instance
        for(size_t i = start; i < end; ++i) {
            vsize_t* instance = new vsize_t(signaturesSize);
            for (size_t j = 0; j < signaturesSize; ++j) {
                (*instance)[j] = static_cast<size_t>(instancesHashValues[i*signaturesSize + j]);
            }
            (*pSignatures)[i] = instance;
        }
        
        start = end+1;
        end = end + windowSize;
    }
    
    hipFree(dev_computedSignaturesPerInstance);
    hipFree(dev_signaturesBlockSize);
    hipFree(dev_featureList);       
    hipFree(dev_computedSignaturesPerInstance);       
           
}