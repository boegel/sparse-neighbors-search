/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutor: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwigs-University Freiburg im Breisgau
**/

#include "nearestNeighborsCuda.h"
#include "kernel.h"
NearestNeighborsCuda::NearestNeighborsCuda() {
    
}
// NearestNeighborsCuda::NearestNeighborsCuda(int* pFeatureList, float* pValuesList,
//                                         int* pSizeOfInstanceList) {
//    mDev_FeatureList = pFeatureList;
//    mDev_ValuesList = pValuesList;
//    mDev_SizeOfInstanceList = pSizeOfInstanceList;
// //    mMaxNnz = pMaxNnz;
// }
NearestNeighborsCuda::~NearestNeighborsCuda() {
    
}

cudaInstance* NearestNeighborsCuda::computeNearestNeighbors(neighborhood* neighbors, size_t pSimilarity, SparseMatrixFloat* pRawData,
                                                                int* pJumpLength) {
    
    
    // int* candidates
    // int* number of candidates per instance
    int* numberOfCandidatesPerInstance = (int*) malloc(neighbors->neighbors->size() * sizeof(int));
    // int* jump length within candidates
    // pJumpLength = (int*) malloc(neighbors->neighbors->size() * sizeof(int));
    int jumpLengthCount = 0;
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        numberOfCandidatesPerInstance[i] = static_cast<int>(neighbors->neighbors->operator[](i).size());
        pJumpLength[i] = jumpLengthCount;
        jumpLengthCount += static_cast<int>(neighbors->neighbors->operator[](i).size());
    }
    int* numberOfCandidatesPerInstanceCuda;
    hipMalloc((void **) &numberOfCandidatesPerInstanceCuda, neighbors->neighbors->size() * sizeof(int));
    hipMemcpy(numberOfCandidatesPerInstanceCuda, numberOfCandidatesPerInstance, neighbors->neighbors->size() * sizeof(int), 
                hipMemcpyHostToDevice);
    int* jumpLengthCuda;
    hipMalloc((void **) &jumpLengthCuda, neighbors->neighbors->size() * sizeof(int));
    hipMemcpy(jumpLengthCuda, pJumpLength, neighbors->neighbors->size() * sizeof(int), 
                hipMemcpyHostToDevice);
    cudaInstance* candidates = (cudaInstance*) malloc(jumpLengthCount * sizeof(cudaInstance));
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        for (size_t j = 0; j < neighbors->neighbors->operator[](i).size(); ++j) {
            candidates[pJumpLength[i] + j].x = neighbors->neighbors->operator[](i)[j];
        }
    }
    
    cudaInstance* candidatesCuda;
    hipMalloc((void **) &candidatesCuda, jumpLengthCount * sizeof(cudaInstance));
    hipMemcpy(candidatesCuda, candidates, jumpLengthCount * sizeof(cudaInstance), 
                hipMemcpyHostToDevice);
  
    if (pSimilarity) {
        // cosineSimilarityCuda<<<32, 96>>>(d_data, size, sizeOfCandidatesCuda, mDev_FeatureList,
        //                                 mDev_ValuesList, mDev_SizeOfInstanceList, mMaxNnz);
    } else {
        euclideanDistanceCuda<<<128, 128>>>(candidatesCuda, jumpLengthCuda, numberOfCandidatesPerInstanceCuda, 
                                        neighbors->neighbors->size(), (*mDev_FeatureList),
                                        (*mDev_ValuesList), (*mDev_SizeOfInstanceList), 
                                        (*mDev_JumpLengthList), (*mDev_DotProducts));
        hipDeviceSynchronize();
    }
    hipMemcpy(candidates, candidatesCuda, sizeof(cudaInstance) * jumpLengthCount, hipMemcpyDeviceToHost);
    hipFree(candidatesCuda);
    free(numberOfCandidatesPerInstance);
    // free(jumpLength);
    hipFree(numberOfCandidatesPerInstanceCuda);
    hipFree(jumpLengthCuda);
    return candidates;
}