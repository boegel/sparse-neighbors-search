/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutor: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwigs-University Freiburg im Breisgau
**/

#include "nearestNeighborsCuda.h"
#include "kernel.h"
NearestNeighborsCuda::NearestNeighborsCuda() {
    
}
// NearestNeighborsCuda::NearestNeighborsCuda(int* pFeatureList, float* pValuesList,
//                                         int* pSizeOfInstanceList) {
//    mDev_FeatureList = pFeatureList;
//    mDev_ValuesList = pValuesList;
//    mDev_SizeOfInstanceList = pSizeOfInstanceList;
// //    mMaxNnz = pMaxNnz;
// }
NearestNeighborsCuda::~NearestNeighborsCuda() {
    
}

cudaInstanceVector* NearestNeighborsCuda::computeNearestNeighbors(neighborhood* neighbors, size_t pSimilarity, SparseMatrixFloat* pRawData) {
    
    
    // int* candidates
    // int* number of candidates per instance
    int* numberOfCandidatesPerInstance = (int*) malloc(neighbors->neighbors->size() * sizeof(int));
    // int* jump length within candidates
    int* jumpLength = (int*) malloc(neighbors->neighbors->size() * sizeof(int));
    int jumpLengthCount = 0;
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        numberOfCandidatesPerInstance[i] = static_cast<int>(neighbors->neighbors->operator[](i).size());
        jumpLength[i] = jumpLengthCount;
        jumpLengthCount = static_cast<int>(neighbors->neighbors->operator[](i).size());
    }
    int* numberOfCandidatesPerInstanceCuda;
    hipMalloc((void **) &numberOfCandidatesPerInstanceCuda, neighbors->neighbors->size() * sizeof(int));
    hipMemcpy(numberOfCandidatesPerInstanceCuda, numberOfCandidatesPerInstance, neighbors->neighbors->size() * sizeof(int), 
                hipMemcpyHostToDevice);
    int* jumpLengthCuda;
    hipMalloc((void **) &jumpLengthCuda, neighbors->neighbors->size() * sizeof(int));
    hipMemcpy(jumpLengthCuda, jumpLength, neighbors->neighbors->size() * sizeof(int), 
                hipMemcpyHostToDevice);
    cudaInstance* candidates = (cudaInstance*) malloc(jumpLengthCount * sizeof(cudaInstance));
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        for (size_t j = 0; j < neighbors->neighbors->operator[](i).size(); ++j) {
            candidates[jumpLength[i] + j].x = neighbors->neighbors->operator[](i)[j];
        }
    }
    
    cudaInstance* candidatesCuda;
    hipMalloc((void **) &candidatesCuda, jumpLengthCount * sizeof(cudaInstance));
    hipMemcpy(candidatesCuda, candidates, jumpLengthCount * sizeof(cudaInstance), 
                hipMemcpyHostToDevice);
    // printf("%u", __LINE__);
    // cudaInstanceVector* candidates;// = (cudaInstanceVector*) malloc(sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    // hipMallocManaged(&candidates, sizeof(cudaInstanceVector) *static_cast<int>(neighbors->neighbors->size()));
    // hipDeviceSynchronize();
    // int* sizeOfCandidates;// = (int*) malloc (sizeof(int) * neighbors->neighbors->size());
    // hipMallocManaged(&sizeOfCandidates, sizeof(int) * static_cast<int>(neighbors->neighbors->size()));
    // hipDeviceSynchronize();
    // for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
    //     hipMallocManaged(&(candidates[i].instance), sizeof(cudaInstance) * static_cast<int>(neighbors->neighbors->operator[](i).size()));
    //     hipDeviceSynchronize();
    //     // candidates[i].instance = (cudaInstance*) malloc(sizeof(cudaInstance) * neighbors->neighbors->operator[](i).size());
    //     sizeOfCandidates[i] = static_cast<int>(neighbors->neighbors->operator[](i).size());
    //     for (size_t j = 0; j < neighbors->neighbors->operator[](i).size(); ++j) {
    //         candidates[i].instance[j].x = static_cast<int>(neighbors->neighbors->operator[](i)[j]);
    //         hipDeviceSynchronize();
    //     }  
    // }
    // printf("%u", __LINE__);
    
    // printf("size %u", neighbors->neighbors->size());
    // cudaInstanceVector* h_data = (cudaInstanceVector*) malloc(sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    // hipMemcpy(d_data, h_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size(), hipMemcpyHostToDevice);
    // printf("h_data[i] list pointer adress: %u\n", candidates[0].instance);
    // printf("h_data[i] list pointer adress: %u\n", &(candidates[0].instance));
    
    // for (size_t i = 0; i < neighbors->neighbors->operator[](0).size(); ++i) {
    //     printf("h_data[i] list pointer adress: %u\n", candidates[0].instance[i].x);
    //     // printf("&h_data[i] list pointer adress: %u\n", &(h_data[i].instance));
    //     // printf("d_data[i] list pointer adress: %u\n", d_data[i]);
    //     // printf("&d_data[i] list pointer adress: %u\n", &(d_data[i]));
    // }
    // memcpy(h_data, candidates, sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    // for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        // printf("size: %u", neighbors->neighbors->operator[](i).size());
        // printf("sizeof(cudaInstance): %u", sizeof(cudaInstance));
        
        // hipMalloc((void **) &(h_data[i].instance), sizeof(cudaInstance) * neighbors->neighbors->operator[](i).size());
        // hipMemcpy(h_data[i].instance, candidates[i].instance, sizeof(cudaInstance) * neighbors->neighbors->operator[](i).size(), hipMemcpyHostToDevice);
    // }
    
    // cudaInstanceVector* d_data;
    // hipMalloc((void **) &d_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    // hipMemcpy(d_data, h_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size(), hipMemcpyHostToDevice);
    // for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
    //     printf("h_data[i] list pointer adress: %u\n", h_data[i].instance);
    //     printf("&h_data[i] list pointer adress: %u\n", &(h_data[i].instance));
    //     // printf("d_data[i] list pointer adress: %u\n", d_data[i]);
    //     // printf("&d_data[i] list pointer adress: %u\n", &(d_data[i]));
    // }
    // int* sizeOfCandidatesCuda;
    // int size = neighbors->neighbors->size();
    // // hipMalloc((void **) &sizeOfCandidatesCuda, sizeof(int) * neighbors->neighbors->size());
    // // hipMemcpy(sizeOfCandidatesCuda, sizeOfCandidates, sizeof(int) * neighbors->neighbors->size(), hipMemcpyHostToDevice);
    
    // printf("Neihgbor Feature list pointer adress: %u\n", (*mDev_FeatureList));
    // printf("Neihgbor &Feature list pointer adress: %u\n", &(*mDev_FeatureList));
    // printf("Neihgbor pDevValueList pointer adress: %u\n", (*mDev_ValuesList));
    // printf("Neihgbor &pDevValueList pointer adress: %u\n", &(*mDev_ValuesList));
    // printf("Neihgbor Inverse size pointer adress: %u\n", (*mDev_SizeOfInstanceList));
    // printf("Neihgbor Inverse &size pointer adress: %u\n", &(*mDev_SizeOfInstanceList));
    // printf("Neihgbor jumppointer adress: %u\n", (*mDev_JumpLengthList));
    // printf("Neihgbor &jump pointer adress: %u\n", &(*mDev_JumpLengthList));
    if (pSimilarity) {
        // cosineSimilarityCuda<<<32, 96>>>(d_data, size, sizeOfCandidatesCuda, mDev_FeatureList,
        //                                 mDev_ValuesList, mDev_SizeOfInstanceList, mMaxNnz);
    } else {
    printf("%u\n", __LINE__);
    fflush(stdout);  
        euclideanDistanceCuda<<<128, 128>>>(candidatesCuda, jumpLengthCuda, numberOfCandidatesPerInstanceCuda, 
                                        neighbors->neighbors->size(), (*mDev_FeatureList),
                                        (*mDev_ValuesList), (*mDev_SizeOfInstanceList), 
                                        (*mDev_JumpLengthList), (*mDev_DotProducts));
        hipDeviceSynchronize();
    printf("%u\n", __LINE__);
    fflush(stdout); 
        
    }
    printf("%u\n", __LINE__);
    fflush(stdout); 
    
    printf("75");
    fflush(stdout);
    
    // hipMemcpy(h_data, d_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size(), hipMemcpyDeviceToHost);
    // printf("78");
    // fflush(stdout);
    
    // for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
    //     hipMemcpy((void **) candidates[i].instance, h_data[i].instance, sizeof(cudaInstance) * neighbors->neighbors->operator[](i).size(), hipMemcpyDeviceToHost);
        // hipFree(candidates[i].instance);
    // }
    // hipFree(candidates)
    // printf("84");
    // fflush(stdout);
    
    // memcpy(candidates, h_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    // hipMemcpy(candidates, d_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size(), hipMemcpyDeviceToHost);
    // printf("88");
    // fflush(stdout);
    
    // hipFree(d_data);
    // cudaInstance* candidatesInstance = (cudaInstance* )
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        // hipMallocManaged(&(candidates[i].instance), static_cast<int>(neighbors->neighbors->operator[](i).size()));
        // hipDeviceSynchronize();
        // candidates2[i].instance = (cudaInstance*) malloc(sizeof(cudaInstance) * neighbors->neighbors->operator[](i).size());
        // sizeOfCandidates[i] = static_cast<int>(neighbors->neighbors->operator[](i).size());
        hipMemcpy(candidates, candidatesCuda, sizeof(cudaInstance) * jumpLengthCount, hipMemcpyDeviceToHost);
        // hipDeviceSynchronize();
        // candidates2[i].instance
        // for (size_t j = 0; j < neighbors->neighbors->operator[](i).size(); ++j) {
        //     candidates2[i].instance[j].x = candidates[i].instance[j].x;
        //     candidates2[i].instance[j].y = candidates[i].instance[j].y;
            
        //     hipDeviceSynchronize();
        // }  
    }
    printf("%u\n", __LINE__);
    fflush(stdout);
    cudaInstanceVector* candidates2 = (cudaInstanceVector*) malloc(sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    // cudaInstance* candidatesLocal;
    printf("%u\n", __LINE__);
    fflush(stdout);
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        // hipMallocManaged(&(candidates[i].instance), sizeof(cudaInstance) * static_cast<int>(neighbors->neighbors->operator[](i).size()));
        // hipDeviceSynchronize();
         candidates2[i].instance = (cudaInstance*) malloc(sizeof(cudaInstance) * neighbors->neighbors->operator[](i).size());
        // sizeOfCandidates[i] = static_cast<int>(neighbors->neighbors->operator[](i).size());
        for (size_t j = 0; j < neighbors->neighbors->operator[](i).size(); ++j) {
            candidates2[i].instance[j].x = candidates[jumpLength[i]+j].x;
            candidates2[i].instance[j].y = candidates[jumpLength[i]+j].y;
            
            // hipDeviceSynchronize();
        }  
    }
    // hipFree(sizeOfCandidates);
    
    // free(h_data);
    // free(sizeOfCandidates);
    printf("94");
    fflush(stdout);
    
    return candidates2;
}