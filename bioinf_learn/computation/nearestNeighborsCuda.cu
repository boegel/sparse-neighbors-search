/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutor: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwigs-University Freiburg im Breisgau
**/

#include "nearestNeighborsCuda.h"
#include "kernel.h"
NearestNeighborsCuda::NearestNeighborsCuda() {
    
}
NearestNeighborsCuda::NearestNeighborsCuda(int* pFeatureList, float* pValuesList,
                                        int* pSizeOfInstanceList, int pMaxNnz) {
   mDev_FeatureList = pFeatureList;
   mDev_ValuesList = pValuesList;
   mDev_SizeOfInstanceList = pSizeOfInstanceList;
   mMaxNnz = pMaxNnz;
}
NearestNeighborsCuda::~NearestNeighborsCuda() {
    
}

cudaInstanceVector* NearestNeighborsCuda::computeNearestNeighbors(neighborhood* neighbors, size_t pSimilarity, const SparseMatrixFloat* pRawData) {
    // hipMalloc((void **) &mDev_FeatureList,
    //         pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(int));
    // // memory for the values of the features of the instances
    // hipMalloc((void **) &mDev_ValuesList, 
    //             pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(float));
    // // memory for the number of features per instance
    // hipMalloc((void **) &mDev_SizeOfInstanceList,
    //         pRawData->getNumberOfInstances() * sizeof(int));
    
    // int* dev_index = (int*) malloc(sizeof(int) * pRawData->getMaxNnz() * pRawData->getNumberOfInstances());
    // for (unsigned int i = 0; i < pRawData->getMaxNnz() * pRawData->getNumberOfInstances(); ++i) {
    //     dev_index[i] = static_cast<int>(pRawData->getSparseMatrixIndex()[i]);
    // }
    // // copy instances and their feature ids to the gpu
    // hipMemcpy(mDev_FeatureList, dev_index,
    //             pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(int),
    //         hipMemcpyHostToDevice);
    // // copy instances and their values for each feature to the gpu
    // float* dev_values = (float*) malloc(sizeof(float) * pRawData->getMaxNnz() * pRawData->getNumberOfInstances());
    // for (unsigned int i = 0; i < pRawData->getMaxNnz() * pRawData->getNumberOfInstances(); ++i) {
    //     dev_values[i] = static_cast<float>(pRawData->getSparseMatrixValues()[i]);
    // }
    // hipMemcpy(mDev_ValuesList, dev_values,
    //             pRawData->getMaxNnz() * pRawData->getNumberOfInstances() * sizeof(float),
    //         hipMemcpyHostToDevice);
            
    // int* dev_sizes = (int*) malloc(sizeof(int) * pRawData->getNumberOfInstances());
    // for (unsigned int i = 0; i < pRawData->getNumberOfInstances(); ++i) {
    //     dev_sizes[i] = static_cast<int>(pRawData->getSparseMatrixSizeOfInstances()[i]);
    // }
    // // copy the size of all instances to the gpu               
    // hipMemcpy(mDev_SizeOfInstanceList, dev_sizes,
    //         pRawData->getNumberOfInstances() * sizeof(int),
    //         hipMemcpyHostToDevice);
    //  for (unsigned int i = 0; i < pRawData->getNumberOfInstances(); ++i) {
    //     // printf ("instanceId: %i, size: %i\n", i, dev_sizes[i]);
    //  }
     
     
    //   for (int i = 0; i < pRawData->getMaxNnz(); ++i) {
    //         // if (i % 100 == 0) {
    //             // for (int j = 0; j < pSizeOfCandidates[i]; ++j) {
    //                 // if (j % 20 == 0) {
    //                     printf ("feature: %i, value: %f\n", dev_index[i], pRawData->getSparseMatrixValues()[i]);
                        
    //                 // }
    //             // }
    //         // }   
    //     }
    // free(dev_index);
    //     free(dev_values);
    //  free(dev_sizes);
   
   
    // printf("30");
    // fflush(stdout);
    cudaInstanceVector* candidates = (cudaInstanceVector*) malloc(sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    int* sizeOfCandidates = (int*) malloc (sizeof(int) * neighbors->neighbors->size());
    // printf("33");
    // fflush(stdout);
    
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        candidates[i].instance = (cudaInstance*) malloc(sizeof(cudaInstance) * neighbors->neighbors->operator[](i).size());
        sizeOfCandidates[i] = neighbors->neighbors->operator[](i).size();
        // if (i % 100 == 0) {
        // printf ("CPUcandidate: %i, size: %i\n", i, sizeOfCandidates[i]);
        // }
        for (size_t j = 0; j < neighbors->neighbors->operator[](i).size(); ++j) {
            candidates[i].instance[j].x = static_cast<int>(neighbors->neighbors->operator[](i)[j]);
            //  if (i % 100 == 0)
            // printf("cpuinstanceid: %i ,size: %i ", candidates[i].instance[j].x, sizeOfCandidates[i]);
            // candidates[i].instance[j].y = neighbors->distances->operator[](i)[j];
        }  
        //  if (i % 100 == 0)
        // printf("\n");      
    }
    // printf("44");
    // fflush(stdout);
    
    // printf("size of neighbirs: %i",neighbors->neighbors->size());
    cudaInstanceVector* h_data = (cudaInstanceVector*) malloc(sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    memcpy(h_data, candidates, sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    // cudaInstanceVector* candidatesCudaHost = (cudaInstanceVector*) malloc(sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        // if (i == 130) {
        // printf("size instance: %i",  neighbors->neighbors->operator[](i).size());
        // }
        hipMalloc((void **) &(h_data[i].instance), sizeof(cudaInstance) * neighbors->neighbors->operator[](i).size());
        hipMemcpy(h_data[i].instance, candidates[i].instance, sizeof(cudaInstance) * neighbors->neighbors->operator[](i).size(), hipMemcpyHostToDevice);
    }
    // printf("57");
    // fflush(stdout);
    
    cudaInstanceVector* d_data;
    hipMalloc((void **) &d_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    hipMemcpy(d_data, h_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size(), hipMemcpyHostToDevice);
    // printf("62");
    // fflush(stdout);
    
    int* sizeOfCandidatesCuda;
    int size = neighbors->neighbors->size();
    hipMalloc((void **) &sizeOfCandidatesCuda, sizeof(int) * neighbors->neighbors->size());
    hipMemcpy(sizeOfCandidatesCuda, sizeOfCandidates, sizeof(int) * neighbors->neighbors->size(), hipMemcpyHostToDevice);
    if (pSimilarity) {
        cosineSimilarityCuda<<<32, 96>>>(d_data, size, sizeOfCandidatesCuda, mDev_FeatureList,
                                        mDev_ValuesList, mDev_SizeOfInstanceList, mMaxNnz);
    } else {
        euclideanDistanceCuda<<<32, 96>>>(d_data, size, sizeOfCandidatesCuda, mDev_FeatureList,
                                        mDev_ValuesList, mDev_SizeOfInstanceList, mMaxNnz);
    }
    // printf("75");
    // fflush(stdout);
    
    // hipMemcpy(h_data, d_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size(), hipMemcpyDeviceToHost);
    // printf("78");
    // fflush(stdout);
    
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        hipMemcpy((void **) candidates[i].instance, h_data[i].instance, sizeof(cudaInstance) * neighbors->neighbors->operator[](i).size(), hipMemcpyDeviceToHost);
        hipFree(h_data[i].instance);
    }
    // printf("84");
    // fflush(stdout);
    
    // memcpy(candidates, h_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size());
    // hipMemcpy(candidates, d_data, sizeof(cudaInstanceVector) * neighbors->neighbors->size(), hipMemcpyDeviceToHost);
    // printf("88");
    // fflush(stdout);
    
    hipFree(d_data);
    hipFree(sizeOfCandidatesCuda);
    free(h_data);
    free(sizeOfCandidates);
    // printf("94");
    // fflush(stdout);
    
    return candidates;
}