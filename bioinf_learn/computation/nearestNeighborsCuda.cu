#include "hip/hip_runtime.h"
/**
 Copyright 2016 Joachim Wolff
 Master Thesis
 Tutor: Fabrizio Costa, Milad Miladi
 Winter semester 2015/2016

 Chair of Bioinformatics
 Department of Computer Science
 Faculty of Engineering
 Albert-Ludwigs-University Freiburg im Breisgau
**/

#include "nearestNeighborsCuda.h"
#include "kernel.h"
#include "typeDefinitionsBasic.h"


NearestNeighborsCuda::NearestNeighborsCuda() {
    
}

NearestNeighborsCuda::~NearestNeighborsCuda() {
    
}

neighborhood* NearestNeighborsCuda::computeNearestNeighbors(neighborhood* neighbors, size_t pSimilarity, SparseMatrixFloat* pRawData,
                                                            SparseMatrixFloat* pOriginalRawData) {
    // if pRawData == null set pointers to original data
    // else load new data to gpu
    //      compute dotProducts
        printf("%i\n", __LINE__);


    float* precomputedDotProductNeighbor;
    int* featureIdsNeighbor;
    float* valuesNeighbor;
    size_t maxNnzNeighbor;
    size_t* sizeNeighbor;
    
    float* precomputedDotProductInstance;
    int* featureIdsInstance;
    float* valuesInstance;
    size_t maxNnzInstance;
    size_t* sizeInstance;
   
    printf("%i\n", __LINE__);
    
    maxNnzNeighbor = pOriginalRawData->getMaxNnz();
    hipMalloc((void **) &precomputedDotProductNeighbor, sizeof(float) * pOriginalRawData->size());
    hipMalloc((void **) &featureIdsNeighbor, sizeof(int) * pOriginalRawData->size() * pOriginalRawData->getMaxNnz());
    hipMalloc((void **) &valuesNeighbor, sizeof(float) * pOriginalRawData->size() * pOriginalRawData->getMaxNnz());
    hipMalloc((void **) &sizeNeighbor, sizeof(size_t) * pOriginalRawData->size());
    
    hipMemcpy(featureIdsNeighbor, pOriginalRawData->getSparseMatrixIndex(),
            pOriginalRawData->size() * pOriginalRawData->getMaxNnz() * sizeof(int),
        hipMemcpyHostToDevice);

    hipMemcpy(valuesNeighbor, pOriginalRawData->getSparseMatrixValues(),
            pOriginalRawData->size() * pOriginalRawData->getMaxNnz() * sizeof(float),
        hipMemcpyHostToDevice);
    hipMemcpy(sizeNeighbor, pOriginalRawData->getSparseMatrixSizeOfInstances(),
        sizeof(size_t) * pOriginalRawData->size(),
        hipMemcpyHostToDevice);  
    // computeDotProducts<<<128, 128>>>(sizeNeighbor);
         
    dotProductSingle<<<128, 128>>>(featureIdsNeighbor, valuesNeighbor, sizeNeighbor, 
                                    pOriginalRawData->size(), pOriginalRawData->getMaxNnz(), precomputedDotProductNeighbor);
   
    if (pRawData == NULL) {
        precomputedDotProductInstance = precomputedDotProductNeighbor;
        featureIdsInstance = featureIdsNeighbor;
        valuesInstance = valuesNeighbor;
        maxNnzInstance = maxNnzNeighbor;
        sizeInstance = sizeNeighbor;
    
    // float* precomputedDotProductNeighbor;
    // int* featureIdsNeighbor;
    // float* valuesNeighbor;
    // size_t maxNnzNeighbor;
    // size_t* sizeNeighbor;
    } else {
       
   
        printf("%i\n", __LINE__);
        
        maxNnzInstance = pRawData->getMaxNnz();
        hipMalloc((void **) &precomputedDotProductInstance, sizeof(float) * pRawData->size());
        hipMalloc((void **) &featureIdsInstance, sizeof(int) * pRawData->size() * pRawData->getMaxNnz());
        hipMalloc((void **) &valuesInstance, sizeof(float) * pRawData->size() * pRawData->getMaxNnz());
        hipMalloc((void **) &sizeInstance, sizeof(size_t) * pRawData->size());
        
        hipMemcpy(featureIdsInstance, pRawData->getSparseMatrixIndex(),
                pRawData->size() * pRawData->getMaxNnz() * sizeof(int),
            hipMemcpyHostToDevice);
    
        hipMemcpy(valuesInstance, pRawData->getSparseMatrixValues(),
                pRawData->size() * pRawData->getMaxNnz() * sizeof(float),
            hipMemcpyHostToDevice);
        hipMemcpy(sizeInstance, pRawData->getSparseMatrixSizeOfInstances(),
            sizeof(size_t) * pRawData->size(),
            hipMemcpyHostToDevice);   
        dotProductSingle<<<128, 128>>>(featureIdsInstance, valuesInstance, sizeInstance, 
                                        pRawData->size(), pRawData->getMaxNnz(), precomputedDotProductInstance);
    }
    // compute dotproducts for all pairs
        printf("%i\n", __LINE__);
    
    size_t* jumpLengthList = (size_t*) malloc(neighbors->neighbors->size() * sizeof(size_t));
    size_t count = 0;
    // size_t jumpLength = 0;
    size_t* candidatesSize = (size_t*) malloc(neighbors->neighbors->size() * sizeof(size_t));
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        jumpLengthList[i] = count;
        count += neighbors->neighbors->operator[](i).size();
        // printf("count: %i\n", count);
       
        candidatesSize[i] = neighbors->neighbors->operator[](i).size();
        // printf("%u: %u\n", i, candidatesSize[i]);
    }
        printf("%i\n", __LINE__);
    
    float3* dotProducts;
        printf("%i\n", __LINE__);
    
    hipMalloc((void **) &dotProducts, sizeof(float3) * count);
        printf("count: %i, %i\n", count, __LINE__);
    
    int* candidates = (int*) malloc(count * sizeof(int));
        printf("%i\n", __LINE__);
        // printf("count: %i\n", count);
        
    
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        for (size_t j = 0; j < neighbors->neighbors->operator[](i).size(); ++j) {
            // printf("jumpLengthList[i]: %i + j: %i\n", jumpLengthList[i], j);
            candidates[jumpLengthList[i]+j] = neighbors->neighbors->operator[](i)[j];
            // printf("%i, ", candidates[jumpLengthList[i]+j]);

        }
        // printf("\n");
    } 
        printf("%i\n", __LINE__);
    
    int* candidatesCuda;
    hipMalloc((void **) &candidatesCuda, count * sizeof(int));
    hipMemcpy(candidatesCuda, candidates, count * sizeof(int), hipMemcpyHostToDevice);
    size_t* jumpLengthListCuda;
    hipMalloc((void **) &jumpLengthListCuda, neighbors->neighbors->size() * sizeof(size_t));
    hipMemcpy(jumpLengthListCuda, jumpLengthList, neighbors->neighbors->size() * sizeof(size_t), hipMemcpyHostToDevice);
    size_t* candidatesSizeCuda;
    hipMalloc((void **) &candidatesSizeCuda, neighbors->neighbors->size() * sizeof(size_t));
    hipMemcpy(candidatesSizeCuda, candidatesSize, neighbors->neighbors->size() * sizeof(size_t), hipMemcpyHostToDevice);
    // call computDotProducts
    printf("%i\n", __LINE__); 
    printf("pMaxNnzNeighbor: %u, pMaxNnzInstance: %u\n", maxNnzNeighbor, maxNnzInstance);
    computeDotProducts<<<128, 128>>>(dotProducts, count, candidatesCuda, jumpLengthListCuda,
                                      candidatesSizeCuda, featureIdsNeighbor, valuesNeighbor,
                                      maxNnzNeighbor, sizeNeighbor,
                                      featureIdsInstance, valuesInstance, maxNnzInstance,
                                      sizeInstance, precomputedDotProductNeighbor, precomputedDotProductInstance);
    
    // computeDotProducts<<<128, 128>>>(sizeNeighbor);
    //    printf("DotproductNeighbors: %u\n", mDev_DotProducts);

    // float* dotCuda = (float*) malloc(4337 * sizeof(float));
    // hipMemcpy(dotCuda, (*mDev_DotProducts), sizeof(float) * 4337, hipMemcpyDeviceToHost);
    // printf("Precomputed dotptoducts: \n");
    // for (size_t  i = 0; i < 4337; ++i) {
    //     printf("%i: %f\n", i, dotCuda[i]);
    // }
    // printf("\n\n"); 
    float* resultsCuda;
    hipMalloc((void **) &resultsCuda, sizeof(float) * count);
    // compute euclidean distance or cosine similarity
    if (pSimilarity) {
        printf("%i\n", __LINE__);
        
    } else {
        printf("%i\n", __LINE__);
        
        euclideanDistanceCuda<<<128, 128>>>(dotProducts, count, resultsCuda);
    }
        printf("%i\n", __LINE__);
    
     // copy data back and sort
    float* results = (float*) malloc( sizeof(float) * count);
        printf("%i\n", __LINE__);
    
    hipMemcpy(results, resultsCuda, sizeof(float) * count, hipMemcpyDeviceToHost);
        printf("%i\n", __LINE__);
    // return results;
    neighborhood* neighbors_ = new neighborhood();;
    neighbors_->neighbors = new vvsize_t(neighbors->neighbors->size());
    neighbors_->distances = new vvfloat(neighbors->neighbors->size());
        printf("%i\n", __LINE__);
    
    for (size_t i = 0; i < neighbors->neighbors->size(); ++i) {
        std::vector<sortMapFloat> returnValue(neighbors->neighbors->operator[](i).size());
        // printf("%i\n", __LINE__);
        
        for (size_t j = 0; j < neighbors->neighbors->operator[](i).size(); ++j) {
            sortMapFloat element; 
            element.key = neighbors->neighbors->operator[](i)[j];
            element.val = results[jumpLengthList[i]+j];
            if (i+j < 32)
            // printf("%f, ", element.val);
            returnValue[j] = element;
        }
        if (i == 0)
        // printf("\n");
        if (pSimilarity) {
            std::sort(returnValue.begin(), returnValue.end(), mapSortDescByValueFloat);
        } else {
            std::sort(returnValue.begin(), returnValue.end(), mapSortAscByValueFloat);
        }
        size_t vectorSize = returnValue.size();
                
        std::vector<size_t> neighborsVector(vectorSize);
        std::vector<float> distancesVector(vectorSize);
        if (vectorSize == 0) {
            neighborsVector.push_back(i);
            distancesVector.push_back(0.0);
        }
        for (size_t j = 0; j < vectorSize; ++j) {
                neighborsVector[j] = returnValue[j].key;
                distancesVector[j] = returnValue[j].val;
        }
        neighbors_->neighbors->operator[](i) = neighborsVector;
        neighbors_->distances->operator[](i) = distancesVector;
    } 
    
    // free memory
    
        printf("%i\n", __LINE__);
   
//    dotProducts, count, candidatesCuda, jumpLengthListCuda,
//                                       candidatesSizeCuda, featureIdsNeighbor, valuesNeighbor,
//                                       maxNnzNeighbor, sizeNeighbor,
//                                       featureIdsInstance, valuesInstance, maxNnzInstance,
//                                       sizeInstance, precomputedDotProductNeighbor, precomputedDotProductInstance
   hipFree(dotProducts);
   hipFree(candidatesCuda);
   hipFree(jumpLengthListCuda);
   hipFree(candidatesSizeCuda);
   hipFree(featureIdsNeighbor);
   hipFree(valuesNeighbor);
   hipFree(sizeNeighbor);
   hipFree(featureIdsInstance);
   hipFree(valuesInstance);
   hipFree(sizeInstance);
   hipFree(precomputedDotProductNeighbor);
   hipFree(precomputedDotProductInstance);
//    hipFree();
   
   
    return neighbors_;
}